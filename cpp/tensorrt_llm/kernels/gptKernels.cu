#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaBf16Wrapper.h"
#include "tensorrt_llm/common/cudaFp8Utils.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/decoderMaskedMultiheadAttentionUtils.h"
#include "tensorrt_llm/kernels/gptKernels.h"
#include <hipcub/hipcub.hpp>

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

// A stateful callback functor that maintains the running sum between consecutive scans.
struct BlockPrefixCallbackOp
{
    // Running prefix
    int mRunningTotal;

    // Constructor
    __device__ BlockPrefixCallbackOp(int runningTotal)
        : mRunningTotal(runningTotal)
    {
    }

    // Thread-0 is responsible for returning a value for seeding the block-wide scan.
    __device__ int operator()(int blockAggregate)
    {
        int oldPrefix = mRunningTotal;
        mRunningTotal += blockAggregate;
        return oldPrefix;
    }
};

// Given an array of sequence lengths, with batchSize elements, that kernel computes the exclusive
// prefix-sums of the sequence lengths. There are (batchSize+1) elements in seqOffsets.
//
// seqOffsets[ 0]        = 0
// seqOffsets[ii]        = seqLengths[0] + .. + seqLengths[ii-1],
// seqOffsets[batchSize] = seqLengths[0] + .. + seqLengths[batchSize-1]
//
// This kernel uses a single thread block of THREADS_PER_BLOCK threads.

// This kernel also computes the padding offsets: Given the index (idx) of a token in a ragged tensor,
// we need the index of the token in the corresponding tensor with padding. We compute an array
// of numTokens elements, called the paddingOffsets, such that the position in the padded tensor
// of the token "idx" in the ragged tensor is given by idx + paddingOffset[idx].
//
// That kernel uses a grid of batchSize blocks.

template <int THREADS_PER_BLOCK, bool COMPUTE_KV_OFFSETS>
__global__ __launch_bounds__(THREADS_PER_BLOCK) void computeSeqAndPaddingOffsets(int* paddingOffsets, int* seqQOffsets,
    int* seqKVOffsets, int const* seqQLengths, int const* seqKVLengths, uint32_t* fmha_tile_counter, int batchSize,
    int maxQSeqLength, bool removePadding, float rotaryEmbeddingScale, float rotaryEmbeddingBase,
    int rotaryEmbeddingDim, RotaryScalingType rotaryScalingType, int rotaryEmbeddingMaxPositions,
    float* rotaryEmbeddingInvFreq, float2* rotaryEmbeddingCoeffCache)
{
    // Dynamic shared memory for storing seqOffsets.
    extern __shared__ int smemSeqQOffsets[];

    // Fixed Q sequence lengths.
    bool const fixed_q_seqlen = seqQLengths == nullptr;

    // The implementation of the parallel scan in the thread block (see CUB for details).
    using BlockScan = hipcub::BlockScan<int, THREADS_PER_BLOCK>;

    // Allocate storage in shared memory to do the scan.
    __shared__ typename BlockScan::TempStorage tempQStorage;
    __shared__ typename BlockScan::TempStorage tempKVStorage;

    // This prefixOp operator keeps a running sum for when we need multiple iterations of the loop.
    BlockPrefixCallbackOp prefixQOp(0);
    BlockPrefixCallbackOp prefixKVOp(0);

    // Iterate over the sequences in the batch.
    //
    // The loop index does not depend on the thread index to make sure all the threads enter the
    // loop as we have __syncthreads in it (and we need all threads to participate to avoid
    // deadlocks).
    // Only the last block computes the full sequence offsets.
    bool const storeSeqOffsets = blockIdx.x == (batchSize - 1);
    int const batchSizeBound = blockIdx.x + 1;
    for (int batchOffset = 0; batchOffset <= batchSizeBound; batchOffset += THREADS_PER_BLOCK)
    {
        // The index of the batch.
        int batchIdx = batchOffset + threadIdx.x;

        // Threads that correspond to valid sequences read the length.
        int seqQLength = 0;
        int seqKVLength = 0;
        if (batchIdx < batchSizeBound)
        {
            seqQLength = fixed_q_seqlen ? maxQSeqLength : seqQLengths[batchIdx];
            if constexpr (COMPUTE_KV_OFFSETS)
            {
                seqKVLength = seqKVLengths[batchIdx];
            }
        }

        // Do the prefix-scan (it calls syncthreads internally).
        int seqQOffset, seqKVOffset;
        BlockScan(tempQStorage).ExclusiveSum(seqQLength, seqQOffset, prefixQOp);
        if constexpr (COMPUTE_KV_OFFSETS)
        {
            BlockScan(tempKVStorage).ExclusiveSum(seqKVLength, seqKVOffset, prefixKVOp);
        }

        // Store the result to smem.
        if (batchIdx <= batchSizeBound)
        {
            smemSeqQOffsets[batchIdx] = seqQOffset;
        }

        // Store the result.
        if (batchIdx <= batchSizeBound && storeSeqOffsets)
        {
            seqQOffsets[batchIdx] = removePadding ? seqQOffset : batchIdx * maxQSeqLength;
            if constexpr (COMPUTE_KV_OFFSETS)
            {
                seqKVOffsets[batchIdx] = seqKVOffset;
            }
        }

        // Make sure the shared memory can be reused for the next iteration of the loop.
        __syncthreads();
    }

    // Compute the padding offsets.
    // Block x dimension is the batch dimension, while threads iterate all tokens in the sequence.
    int batchIdx = blockIdx.x;
    // The beginning of the sequence.
    int seqBegin = smemSeqQOffsets[batchIdx];
    // The offset to the 1st element of the next sequence.
    int seqEnd = smemSeqQOffsets[batchIdx + 1];
    // The length of the sequence.
    int seqLength = seqEnd - seqBegin;

    // The number of padded tokens in the previous sequences.
    int paddingOffset = batchIdx * maxQSeqLength - seqBegin;
    bool const need_padding_offsets = paddingOffsets != nullptr;

    if (need_padding_offsets)
    {
        // Iterate over the tokens to update the number of padded elements.
        for (int tokenIdx = threadIdx.x; tokenIdx < seqLength; tokenIdx += blockDim.x)
        {
            paddingOffsets[seqBegin + tokenIdx] = paddingOffset;
        }
    }

    // Each block generates the rotary embedding inv_freq tensor for the corresponding sequence.
    int zid = 2 * threadIdx.x;
    int halfRotaryEmbeddingDim = rotaryEmbeddingDim / 2;
    if (rotaryEmbeddingDim > 0 && zid < rotaryEmbeddingDim)
    {
        mmha::update_rotary_base_n_scale(rotaryEmbeddingBase, rotaryEmbeddingScale, rotaryScalingType,
            rotaryEmbeddingDim, rotaryEmbeddingMaxPositions, seqKVLengths[batchIdx]);
        float const invFreq = rotaryEmbeddingScale / powf(rotaryEmbeddingBase, zid / (float) rotaryEmbeddingDim);
        rotaryEmbeddingInvFreq[batchIdx * halfRotaryEmbeddingDim + threadIdx.x] = invFreq;
    }

    // Reset fmha tile counter to 0 before launching fmha kernels.
    if (threadIdx.x == 0 && blockIdx.x == 0 && fmha_tile_counter != nullptr)
    {
        fmha_tile_counter[0] = 0u;
    }
}

// This kernel computes the attention mask. We must compute this on-the-fly in the future.

template <typename AttentionMaskDataType>
__global__ void computeAttentionMask(AttentionMaskDataType* attentionMask, int const* seqLengths, int maxQSeqLength,
    int attentionWindowSize, AttentionMaskType attentionMaskType)
{
    // The index of the sequence in the batch.
    int batchIdx = blockIdx.y;

    // The number of items in the mask for each sequence.
    int maskSize = maxQSeqLength * maxQSeqLength;
    // The offset to the 1st element of the mask for that particular sequence.
    int batchOffset = batchIdx * maskSize;

    // The length of the sequence.
    int seqLength = seqLengths[batchIdx];

    // Iterate over the tokens to update the number of padded elements.
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < maskSize; idx += gridDim.x * blockDim.x)
    {
        // The position in the matrix.
        int rowIdx = idx / maxQSeqLength;
        int colIdx = idx % maxQSeqLength;

        // Is it a valid token?
        bool isValid = true;
        switch (attentionMaskType)
        {
        case AttentionMaskType::PADDING:
            isValid = rowIdx < seqLength && colIdx < seqLength;
            // seq_length==4, max_seq_len==5
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 0 0 0 0 0
            break;
        case AttentionMaskType::CAUSAL:
            isValid = rowIdx < seqLength && colIdx < seqLength && colIdx <= rowIdx;
            // Sliding_window_causal when there are not enough kv cache.
            isValid = isValid && colIdx >= max(0, rowIdx - attentionWindowSize);
            // seq_length==4, max_seq_len==5
            // 1 0 0 0 0
            // 1 1 0 0 0
            // 1 1 1 0 0
            // 1 1 1 1 0
            // 0 0 0 0 0

            // seq_length==6, max_seq_len==6, max_attention_window_size = 2
            // 1 0 0 0 0 0
            // 1 1 0 0 0 0
            // 1 1 1 0 0 0
            // 0 1 1 1 0 0
            // 0 0 1 1 1 0
            // 0 0 0 1 1 1
            break;
        case AttentionMaskType::BIDIRECTIONAL:
            // clang-format off
             isValid = (rowIdx <  seqLength - 1 && colIdx < seqLength - 1) ||
                       (rowIdx == seqLength - 1 && colIdx < seqLength);
            // clang-format on
            // seq_length==4, max_seq_len==5
            // 1 1 1 0 0
            // 1 1 1 0 0
            // 1 1 1 0 0
            // 1 1 1 1 0
            // 0 0 0 0 0
        case AttentionMaskType::BIDIRECTIONALGLM:
            // clang-format off
             isValid = (colIdx < seqLength - 1) ||
                       (rowIdx == seqLength - 1 && colIdx == seqLength - 1);
            // clang-format on
            // seq_length==4, max_seq_len==5
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 1 1 1 1 1
            break;
        }

        // Store the mask.
        attentionMask[batchOffset + idx] = isValid ? AttentionMaskDataType(1.f) : AttentionMaskDataType(0.f);
    }
}

template <typename T>
void invokeBuildDecoderInfo(BuildDecoderInfoParams<T> const& params, hipStream_t stream)
{
    // Compute the sequence and padding offsets.
    int const THREADS_PER_BLOCK = 256;
    TLLM_CHECK_WITH_INFO(params.rotaryEmbeddingDim / 2 <= 256 && params.rotaryEmbeddingDim % 2 == 0,
        "Rotary embedding dim is assumed to be smaller than 512 and multiple of 2.");
    TLLM_CHECK_WITH_INFO(
        !(params.seqKVLengths == nullptr && params.rotaryEmbeddingDim > 0), "KV sequence lengths buffer is invalid.");
    const size_t smem_size = (params.batchSize + 1) * sizeof(int);
    if (params.seqKVOffsets)
    {
        TLLM_CHECK_WITH_INFO(params.seqKVLengths != nullptr, "KV sequence lengths buffer is invalid.");
        computeSeqAndPaddingOffsets<THREADS_PER_BLOCK, true>
            <<<params.batchSize, THREADS_PER_BLOCK, smem_size, stream>>>(params.paddingOffsets, params.seqQOffsets,
                params.seqKVOffsets, params.seqQLengths, params.seqKVLengths, params.fmhaTileCounter, params.batchSize,
                params.maxQSeqLength, params.removePadding, params.rotaryEmbeddingScale, params.rotaryEmbeddingBase,
                params.rotaryEmbeddingDim, params.rotaryScalingType, params.rotaryEmbeddingMaxPositions,
                params.rotaryEmbeddingInvFreq, params.rotaryEmbeddingCoeffCache);
    }
    else
    {
        computeSeqAndPaddingOffsets<THREADS_PER_BLOCK, false>
            <<<params.batchSize, THREADS_PER_BLOCK, smem_size, stream>>>(params.paddingOffsets, params.seqQOffsets,
                params.seqKVOffsets, params.seqQLengths, params.seqKVLengths, params.fmhaTileCounter, params.batchSize,
                params.maxQSeqLength, params.removePadding, params.rotaryEmbeddingScale, params.rotaryEmbeddingBase,
                params.rotaryEmbeddingDim, params.rotaryScalingType, params.rotaryEmbeddingMaxPositions,
                params.rotaryEmbeddingInvFreq, params.rotaryEmbeddingCoeffCache);
    }

    // Compute the attention mask, if needed.
    if (params.attentionMask != nullptr)
    {
        TLLM_CHECK_WITH_INFO(params.seqQLengths != nullptr, "Q sequence lengths buffer is invalid.");
        int const MIN_BLOCKS = 512;
        int blocksPerSeq = 16;
        while (blocksPerSeq * params.batchSize < MIN_BLOCKS)
        {
            blocksPerSeq *= 2;
        }
        dim3 grid(blocksPerSeq, params.batchSize);
        computeAttentionMask<<<grid, THREADS_PER_BLOCK, 0, stream>>>(params.attentionMask, params.seqQLengths,
            params.maxQSeqLength, params.attentionWindowSize, params.attentionMaskType);
    }
}

template void invokeBuildDecoderInfo(BuildDecoderInfoParams<float> const&, hipStream_t);
template void invokeBuildDecoderInfo(BuildDecoderInfoParams<half> const&, hipStream_t);
#ifdef ENABLE_BF16
template void invokeBuildDecoderInfo(BuildDecoderInfoParams<__hip_bfloat16> const&, hipStream_t);
#endif
#ifdef ENABLE_FP8
template void invokeBuildDecoderInfo(BuildDecoderInfoParams<__hip_fp8_e4m3_fnuz> const&, hipStream_t);
#endif

__global__ void updatePaddingCountKernel(int* paddingPerSeq, int const* seqLengths, int maxQSeqLength, int batchSize)
{

    for (int ii = threadIdx.x; ii < batchSize; ii += blockDim.x)
    {
        paddingPerSeq[ii] = maxQSeqLength - seqLengths[ii];
    }
}

} // namespace kernels
} // namespace tensorrt_llm
