#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include <stdio.h>

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

__global__ void curandInitialize(hiprandState_t* state, int const* batchSlots, int const size, const uint64_t randomSeed)
{
    int const idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size)
    {
        auto const batchSlot = batchSlots != nullptr ? batchSlots[idx] : idx;
        hiprand_init(randomSeed, 0, 0, &state[batchSlot]);
    }
}

void invokeCurandInitialize(
    hiprandState_t* state, int const* batchSlots, const size_t batchSize, const uint64_t randomSeed, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid((int) (ceil(batchSize * 1.0 / 256)));
    curandInitialize<<<grid, block, 0, stream>>>(state, batchSlots, batchSize, randomSeed);
}

__global__ void curandBatchInitialize(
    hiprandState_t* states, int const* batchSlots, int const size, uint64_t const* randomSeeds)
{
    int const idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size)
    {
        auto const batchSlot = batchSlots != nullptr ? batchSlots[idx] : idx;
        hiprand_init(randomSeeds[batchSlot], 0, 0, &states[batchSlot]);
    }
}

void invokeCurandBatchInitialize(hiprandState_t* states, int const* batchSlots, const size_t batchSize,
    uint64_t const* randomSeeds, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid((int) (ceil(batchSize * 1.0 / 256)));
    curandBatchInitialize<<<grid, block, 0, stream>>>(states, batchSlots, batchSize, randomSeeds);
}

template <typename T>
__global__ void addBiasSoftMax(T* logits, T** logitsPtrs, T* probs, T const* bias, int32_t const* endIds,
    FinishedState const* finished, int32_t const* batchSlots, int32_t batchSize, int32_t maxBatchSize,
    int32_t beamWidth, int32_t vocabSize, int32_t vocabSizePadded, bool skipSoftMax, bool batchSlotsLogits)
{
    auto const batchIdx = blockIdx.x;
    auto const beamIdx = blockIdx.y;
    auto const batchSlot = batchSlots != nullptr ? batchSlots[batchIdx] : batchIdx;
    auto const batchIdxLogits = batchSlotsLogits ? batchSlot : batchIdx;
    FinishedState const finishState
        = finished != nullptr ? finished[beamIdx * maxBatchSize + batchSlot] : FinishedState::empty();
    if (finishState.isSkipDecoding())
    {
        return;
    }

    auto logitsPtr = logitsPtrs ? logitsPtrs[batchIdx] + beamIdx * vocabSizePadded
                                : logits + (batchIdxLogits * beamWidth + beamIdx) * vocabSizePadded;

    bool finish = finishState.isFinished();
    int offset = (batchIdxLogits * beamWidth + beamIdx) * vocabSizePadded;

    float maxVal = -1 * FLT_MAX;
    bool const IS_FP16 = std::is_same<T, half>::value;
    T const MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;
    __shared__ float sMaxVal;
    __shared__ float sSumVal;

    for (int tid = threadIdx.x; tid < vocabSizePadded; tid += blockDim.x)
    {
        auto logit = logitsPtr[tid];
        if (tid < vocabSize)
        {
            if (finish && endIds != nullptr)
            {
                logit = (tid == endIds[batchSlot]) ? MAX_T_VAL : -MAX_T_VAL;
            }
            else
            {
                T bias_val = (bias != nullptr) ? bias[tid] : (T) 0.0f;
                logit += bias_val;
            }
        }
        else
        {
            logit = -MAX_T_VAL;
        }
        maxVal = max(maxVal, (float) logit);
        logitsPtr[tid] = logit;
    }

    if (!skipSoftMax)
    {
        maxVal = blockReduceMax<float>((float) maxVal);
        if (threadIdx.x == 0)
        {
            sMaxVal = maxVal;
        }
        __syncthreads();

        float sumVal = 0.0f;
        for (int tid = threadIdx.x; tid < vocabSizePadded; tid += blockDim.x)
        {
            probs[offset + tid] = __expf((float) logitsPtr[tid] - sMaxVal);
            sumVal += (float) probs[offset + tid];
        }

        sumVal = blockReduceSum<float>(sumVal);
        if (threadIdx.x == 0)
        {
            sSumVal = sumVal;
        }
        __syncthreads();

        for (int tid = threadIdx.x; tid < vocabSizePadded; tid += blockDim.x)
        {
            probs[offset + tid] = ((float) probs[offset + tid] / (sSumVal + 1e-6f));
        }
    }
}

template <typename T>
void invokeAddBiasSoftMax(T* logits, T** logitsPtrs, T* probs, T const* bias, int32_t const* endIds,
    FinishedState const* finished, int32_t const* batchSlots, int32_t batchSize, int32_t maxBatchSize,
    int32_t beamWidth, int32_t vocabSize, int32_t vocabSizePadded, bool skipSoftMax, bool batchSlotsLogits,
    hipStream_t stream)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    dim3 grid(batchSize, beamWidth);
    auto const vocabRoundedToWarp = roundUp(vocabSize, 32);
    dim3 block(min(vocabRoundedToWarp, 1024));
    // vocabSize, e.g., 30000, 7000.... vocabSize is usually very big.
    addBiasSoftMax<<<grid, block, 0, stream>>>(logits, logitsPtrs, probs, bias, endIds, finished, batchSlots, batchSize,
        maxBatchSize, beamWidth, vocabSize, vocabSizePadded, skipSoftMax, batchSlotsLogits);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template void invokeAddBiasSoftMax(float* logits, float** logitsPtrs, float* probs, float const* bias,
    int32_t const* endIds, FinishedState const* finished, int32_t const* batchSlots, int32_t batchSize,
    int32_t maxBatchSize, int32_t beamWidth, int32_t vocabSize, int32_t vocabSizePadded, bool skipSoftMax,
    bool batchSlotsLogits, hipStream_t stream);

template void invokeAddBiasSoftMax(half* logits, half** logitsPtrs, half* probs, half const* bias,
    int32_t const* endIds, FinishedState const* finished, int32_t const* batchSlots, int32_t batchSize,
    int32_t maxBatchSize, int32_t beamWidth, int32_t vocabSize, int32_t vocabSizePadded, bool skipSoftMax,
    bool batchSlotsLogits, hipStream_t stream);

template <typename T>
__global__ void scatterDecodingParamsKernel(T const* src, T* dst, int const* batchSlots, int batchSize)
{
    auto const batchIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (batchIdx >= batchSize)
    {
        return;
    }
    auto const batchSlot = batchSlots == nullptr ? batchIdx : batchSlots[batchIdx];
    dst[batchSlot] = src[batchIdx];
}

template <typename T>
void invokeScatterDecodingParams(T const* src, T* dst, int const* batchSlots, int batchSize, hipStream_t stream)
{
    constexpr int THREADS_PER_CTA = 256;
    dim3 grid(divUp(batchSize, THREADS_PER_CTA));
    scatterDecodingParamsKernel<<<grid, THREADS_PER_CTA, 0, stream>>>(src, dst, batchSlots, batchSize);
}

template void invokeScatterDecodingParams(
    float const* src, float* dst, int const* batchSlots, int batchSize, hipStream_t stream);
template void invokeScatterDecodingParams(
    uint32_t const* src, uint32_t* dst, int const* batchSlots, int batchSize, hipStream_t stream);
template void invokeScatterDecodingParams(
    int32_t const* src, int32_t* dst, int const* batchSlots, int batchSize, hipStream_t stream);
} // namespace kernels
} // namespace tensorrt_llm
