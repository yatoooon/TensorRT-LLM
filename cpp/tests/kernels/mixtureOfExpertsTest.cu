#include "hip/hip_runtime.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/cutlass_kernels/cutlass_preprocessors.h"
#include "tensorrt_llm/runtime/cudaStream.h"

#include <algorithm>
#include <gtest/gtest.h>
#include <numeric>

#include "tensorrt_llm/kernels/mixtureOfExperts/moe_kernels.h"
#include "tensorrt_llm/runtime/bufferManager.h"

using namespace tensorrt_llm::kernels;
using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

constexpr static float FP8_MAX = 440; // FP8_E4M3_MAX;

__host__ __device__ constexpr float expertShift(int expert, int num_experts)
{
    return float(expert) / num_experts;
}

template <class T>
__global__ void initWeightsKernel(T* data, int64_t w, int64_t h, float base, float scale)
{
    size_t expert_id = blockIdx.z;
    T* start_offset = data + expert_id * w * h;
    float expert_shift = scale * expertShift(expert_id, gridDim.z);

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < w && y < h)
    {
        start_offset[y * w + x] = (x == y) ? T(base * scale + expert_shift) : T(0.f);
    }
}

template <class T>
__global__ void initWeightsGatedKernel(T* data, int64_t w, int64_t h, float base_1, float base_2, float scale)
{
    size_t expert_id = blockIdx.z;
    T* start_offset = data + expert_id * w * h * 2;

    float expert_shift = scale * expertShift(expert_id, gridDim.z);

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < w && y < h)
    {
        start_offset[y * w + x] = (x == y) ? T(base_1 * scale + expert_shift) : T(0.f);
        start_offset[(y + h) * w + x] = (x == y) ? T(base_2 * scale + expert_shift) : T(0.f);
    }
}

template <class T>
__global__ void initBiasToExpertIdKernel(T* data, int64_t w)
{
    size_t expert_id = blockIdx.y;
    T* start_offset = data + expert_id * w;

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < w)
        start_offset[x] = T(expert_id);
}

template <class T>
__global__ void initBiasToExpertIdGatedKernel(T* data, int64_t w)
{
    size_t expert_id = blockIdx.y;
    T* start_offset = data + expert_id * w * 2;

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < w)
    {
        start_offset[x] = T(expert_id);
        start_offset[x + w] = T(expert_id + 1);
    }
}

#ifdef ENABLE_FP8
using SafeFP8 = __hip_fp8_e4m3_fnuz;
#else
using SafeFP8 = void;
#endif

template <class TypeTuple_>
class MixtureOfExpertsTest : public ::testing::Test
{
protected:
    using DataType = typename TypeTuple_::DataType;
    using WeightType = typename TypeTuple_::WeightType;
    using OutputType = typename TypeTuple_::OutputType;
    constexpr static bool INT4 = std::is_same_v<WeightType, cutlass::uint4b_t>;
    constexpr static bool FP8 = std::is_same_v<DataType, SafeFP8>;
    constexpr static bool INT_QUANT = !std::is_same_v<DataType, WeightType>;
    using WeightStorage = std::conditional_t<INT_QUANT, uint8_t, WeightType>;
    constexpr static int WEIGHT_ELEM_PER_BYTE = INT4 ? 2 : 1;
    constexpr static int64_t HIDDEN_SIZE_MULTIPLIER = 8;
    constexpr static int64_t DEFAULT_HIDDEN_SIZE
        = HIDDEN_SIZE_MULTIPLIER * 64 / sizeof(WeightType) * WEIGHT_ELEM_PER_BYTE;

    static BufferManager::CudaStreamPtr mStream;
    static std::unique_ptr<BufferManager> mBufferManager;
    static int mDeviceCount;

    std::vector<BufferManager::IBufferPtr> managed_buffers;
    float* mInputProbabilities{};
    DataType* mInputTensor{};

    int64_t mHiddenSize{};
    int64_t mNumExperts{};
    int64_t mK{};

    float getTolerance(float scale = 1.f)
    {
        // These FP8 tolerances are tuned quite tightly so should pick up any regressions
        // Whether the current results are as tight as they should be requires further investigation
        // They can be much tighter if we use the same value for all experts (so the scaling factors are trivial)
        // But that is hardly representative
        bool loose_fp8 = mIsGated || mNormMode == MOEExpertScaleNormalizationMode::RENORMALIZE;
        float tol = std::is_same_v<DataType, float> ? 0.001
            : std::is_same_v<DataType, half>        ? 0.01
            : std::is_same_v<DataType, SafeFP8>     ? (loose_fp8 ? 0.1 : 0.07)
                                                    : 0.1;

        // Keep the scale in a sane range
        scale = std::clamp(scale, 1.f, 30.f);
        return scale * tol;
    }

    static bool shouldSkip()
    {
#ifndef ENABLE_FP8
        static_assert(!FP8, "FP8 Tests enabled on unsupported CUDA version");
#endif
        bool should_skip_no_device = mDeviceCount <= 0;
        bool should_skip_unsupported_fp8 = getSMVersion() < 90 && FP8;
        return should_skip_no_device || should_skip_unsupported_fp8;
    }

    static void SetUpTestCase()
    {
        mDeviceCount = getDeviceCount();
        if (shouldSkip())
        {
            GTEST_SKIP() << "Skipping due to no/unsupported GPU";
        }

        mStream = std::make_shared<CudaStream>();
        mBufferManager = std::make_unique<BufferManager>(mStream);
    }

    static void TearDownTestCase()
    {
        mBufferManager.reset();
        mStream.reset();
    }

    void SetUp() override
    {
        assert(mBufferManager);
        if (shouldSkip())
        {
            GTEST_SKIP() << "Skipping due to no/unsupported GPU";
        }
    }

    void TearDown()
    {
        managed_buffers.clear();
    }

    void initWeights(DataType* buffer, int64_t w, int64_t h, float base, float scalar)
    {
        dim3 block(16, 16, 1);
        dim3 grid(divUp(w, block.x), divUp(h, block.y), mNumExperts);
        initWeightsKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w, h, base, scalar);
    }

    void initBias(DataType* buffer, int64_t w)
    {
        dim3 block(256, 1, 1);
        dim3 grid(divUp(w, block.x), mNumExperts);
        initBiasToExpertIdKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w);
    }

    void initWeightsGated(DataType* buffer, int64_t w, int64_t h, float base_1, float base_2, float scalar)
    {
        if (!mIsGated)
            return initWeights(buffer, w, h, base_1, scalar);

        h /= 2;
        dim3 block(16, 16, 1);
        dim3 grid(divUp(w, block.x), divUp(h, block.y), mNumExperts);
        initWeightsGatedKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w, h, base_1, base_2, scalar);
    }

    void initBiasGated(DataType* buffer, int64_t w)
    {
        if (!mIsGated)
            return initBias(buffer, w);

        w /= 2;
        dim3 block(256, 1, 1);
        dim3 grid(divUp(w, block.x), mNumExperts);
        initBiasToExpertIdGatedKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w);
    }

    CutlassMoeFCRunner<DataType, WeightType, OutputType> mMoERunner{};
    char* mWorkspace{};
    float* mScaleProbs{};
    DataType* mRawExpertWeight1{};
    DataType* mRawExpertWeight2{};
    WeightStorage* mExpertWeight1{};
    WeightStorage* mExpertWeight2{};
    DataType* mExpertIntScale1{};
    DataType* mExpertIntScale2{};

    float mFP8WeightScalar1{1.f};
    float mFP8WeightScalar2{1.f};
    float* mExpertFP8Scale1{};
    float* mExpertFP8Scale2{};
    float* mExpertFP8Scale3{};

    DataType* mExpertBias1{};
    DataType* mExpertBias2{};

    void* mTpExpertScratch{}; // Copy the experts here when slicing up inputs
    size_t mTpExpertScratchSize{};

    OutputType* mFinalOutput{};
    int* mSourceToExpandedMap;
    int* mSelectedExpert;
    bool* mFinished{};
    int64_t mInterSize{};
    int64_t mTotalTokens{};
    int64_t mActiveRows{};

    bool mUseBias = true;

    bool mIsGated = false;
    int64_t mGatedMultiplier = 1;

    tensorrt_llm::ActivationType mActType = tensorrt_llm::ActivationType::Relu;
    MOEExpertScaleNormalizationMode mNormMode = MOEExpertScaleNormalizationMode::NONE;

    std::optional<tensorrt_llm::cutlass_extensions::CutlassGemmConfig> mSelectedConfig = std::nullopt;

    // Keep to simple power of two so we can have tight bounds on precision for quantized modes
    float const mExpertWDiag1{0.5};
    float const mExpertWDiagGated{1};
    float const mExpertWDiag2{2};

    float mMaxInput{};

    template <class T>
    T* allocBuffer(size_t size)
    {
        managed_buffers.emplace_back(mBufferManager->managed(size * sizeof(T)));
        EXPECT_EQ(hipGetLastError(), hipSuccess) << "Error allocating buffer of size: " << size;
        T* ptr = static_cast<T*>(managed_buffers.back()->data());
        return ptr;
    }

    bool checkSufficientTestMemory(int64_t num_tokens, int64_t hidden_size, int64_t num_experts, int64_t k)
    {
        this->managed_buffers.clear();             // Make sure all the previous buffers are freed
        check_cuda_error(hipDeviceSynchronize()); // Sync to make sure all previous operations are resolved

        size_t weight_size = hidden_size * hidden_size * 4 * num_experts * sizeof(WeightType);
        // Skip the test if the GPU does not have enough memory
        size_t workspace_size = this->mMoERunner.getWorkspaceSize(
            num_tokens, hidden_size, hidden_size * 4, num_experts, k, this->mActType, {});

        size_t total_size = workspace_size + weight_size * 2;

        auto const [freeMem, totalMem] = tensorrt_llm::common::getDeviceMemoryInfo(false);
        return freeMem >= total_size;
    }

    void initBuffersPermute(std::vector<std::vector<DataType>> h_hidden_states,
        std::vector<std::vector<float>> h_router_results, int64_t hidden_size, int64_t num_experts, int64_t k,
        std::vector<uint8_t> finished, MOEParallelismConfig parallelism_config)
    {
        managed_buffers.clear();

        mHiddenSize = hidden_size;
        mInterSize = hidden_size * 4;
        mNumExperts = num_experts;
        mK = k;
        mIsGated = tensorrt_llm::isGatedActivation(mActType);
        mGatedMultiplier = mIsGated ? 2 : 1;
        auto const gated_inter = mInterSize * mGatedMultiplier;

        mTotalTokens = 0;

        std::vector<int64_t> h_seq_lens;
        h_seq_lens.push_back(0);
        for (auto& sequence : h_hidden_states)
        {
            assert(sequence.size() % hidden_size == 0);
            int64_t num_tokens = sequence.size() / hidden_size;
            h_seq_lens.emplace_back(h_seq_lens.back() + num_tokens);
            mTotalTokens += num_tokens;
        }

        size_t workspace_size = mMoERunner.getWorkspaceSize(
            mTotalTokens, mHiddenSize, mInterSize, mNumExperts, mK, mActType, parallelism_config);

        auto const stream = mStream->get();

        mWorkspace = allocBuffer<char>(workspace_size);
        // Memset to an obviously incorrect value, so we detect any issues with uninitialised fields
        check_cuda_error(hipMemsetAsync(mWorkspace, 0xD5, workspace_size, stream));
        size_t const expert_matrix_size = mNumExperts * mHiddenSize * mInterSize;

        mRawExpertWeight1 = allocBuffer<DataType>(expert_matrix_size * mGatedMultiplier);
        mRawExpertWeight2 = allocBuffer<DataType>(expert_matrix_size);

        mTpExpertScratchSize = expert_matrix_size * mGatedMultiplier / parallelism_config.tp_size;
        mTpExpertScratchSize += expert_matrix_size / parallelism_config.tp_size;

        mExpertBias1 = nullptr;
        mExpertBias2 = nullptr;
        if (mUseBias)
        {
            // Allow space for the slice of bias1 in the scratch
            mTpExpertScratchSize += mNumExperts * gated_inter / parallelism_config.tp_size;
            mExpertBias1 = allocBuffer<DataType>(mNumExperts * gated_inter);
            mExpertBias2 = allocBuffer<DataType>(mNumExperts * mHiddenSize);

            check_cuda_error(hipMemsetAsync(mExpertBias1, 0x0, mNumExperts * gated_inter * sizeof(DataType), stream));
            check_cuda_error(hipMemsetAsync(mExpertBias2, 0x0, mNumExperts * mHiddenSize * sizeof(DataType), stream));
        }

        if constexpr (INT_QUANT)
        {
            mExpertWeight1 = allocBuffer<WeightStorage>(expert_matrix_size * mGatedMultiplier / WEIGHT_ELEM_PER_BYTE);
            mExpertWeight2 = allocBuffer<WeightStorage>(expert_matrix_size / WEIGHT_ELEM_PER_BYTE);

            mTpExpertScratchSize += mNumExperts * gated_inter / parallelism_config.tp_size;
            mExpertIntScale1 = allocBuffer<DataType>(mNumExperts * gated_inter);
            mExpertIntScale2 = allocBuffer<DataType>(mNumExperts * mHiddenSize);
        }
        else
        {
            mExpertWeight1 = mRawExpertWeight1;
            mExpertWeight2 = mRawExpertWeight2;
        }

        if constexpr (FP8)
        {
            mExpertFP8Scale1 = allocBuffer<float>(mNumExperts);
            mExpertFP8Scale2 = allocBuffer<float>(1);
            mExpertFP8Scale3 = allocBuffer<float>(mNumExperts);

            EXPECT_NE(mMaxInput, 0.0f);
            initFP8Scales(mMaxInput);
        }

        mTpExpertScratch = nullptr;
        if (parallelism_config.tp_size > 1)
        {
            mTpExpertScratch = allocBuffer<DataType>(mTpExpertScratchSize);
        }

        mActiveRows = mTotalTokens;
        mFinished = nullptr;
        if (!finished.empty())
        {
            mFinished = allocBuffer<bool>(mTotalTokens);
            check_cuda_error(hipMemcpyAsync(
                mFinished, finished.data(), mTotalTokens * sizeof(bool), hipMemcpyHostToDevice, stream));
            static_assert(sizeof(bool) == sizeof(uint8_t), "Test assumes bool is interchangeable with uint8_t");
            mActiveRows = std::count(finished.begin(), finished.end(), 0);
        }

        mInputProbabilities = allocBuffer<float>(mTotalTokens * mNumExperts);
        mScaleProbs = allocBuffer<float>(mTotalTokens * mK);
        mInputTensor = allocBuffer<DataType>(mTotalTokens * mHiddenSize);
        mFinalOutput = allocBuffer<OutputType>(mTotalTokens * mHiddenSize);

        mSourceToExpandedMap = allocBuffer<int>(mTotalTokens * mK);
        mSelectedExpert = allocBuffer<int>(mTotalTokens * mK);

        auto* input_probs_ptr = mInputProbabilities;
        for (auto& sequence : h_router_results)
        {
            check_cuda_error(hipMemcpyAsync(
                input_probs_ptr, sequence.data(), sequence.size() * sizeof(float), hipMemcpyHostToDevice, stream));
            input_probs_ptr += sequence.size();
        }

        auto* hidden_states_ptr = mInputTensor;
        for (auto& sequence : h_hidden_states)
        {
            check_cuda_error(hipMemcpyAsync(hidden_states_ptr, sequence.data(), sequence.size() * sizeof(DataType),
                hipMemcpyHostToDevice, stream));
            hidden_states_ptr += sequence.size();
        }

        check_cuda_error(hipStreamSynchronize(stream));

        // Init the diagonals of our matrix, this will set to the scalar value
        initWeightsGated(
            mRawExpertWeight1, mHiddenSize, gated_inter, mExpertWDiag1, mExpertWDiagGated, mFP8WeightScalar1);
        initWeights(mRawExpertWeight2, mInterSize, mHiddenSize, mExpertWDiag2, mFP8WeightScalar2);

        if (mUseBias)
        {
            initBiasGated(mExpertBias1, gated_inter);
            initBias(mExpertBias2, mHiddenSize);
        }

        if constexpr (INT_QUANT)
        {
            cutlass_kernels::QuantType quant_type
                = INT4 ? cutlass_kernels::QuantType::W4_A16 : cutlass_kernels::QuantType::W8_A16;

            std::vector<size_t> shape1{(size_t) mNumExperts, (size_t) mHiddenSize, (size_t) gated_inter};
            std::vector<size_t> shape2{(size_t) mNumExperts, (size_t) mInterSize, (size_t) mHiddenSize};

            doIntQuant(quant_type, shape1, mRawExpertWeight1, mExpertIntScale1, mExpertWeight1);
            doIntQuant(quant_type, shape2, mRawExpertWeight2, mExpertIntScale2, mExpertWeight2);
        }

        check_cuda_error(hipStreamSynchronize(stream));
    }

    void doIntQuant(cutlass_kernels::QuantType quant_type, std::vector<size_t> shape, DataType* inputs,
        DataType* scales, uint8_t* outputs)
    {
        // Runs on the CPU, must be after stream sync
        if constexpr (INT_QUANT)
        {
            size_t elems = std::reduce(shape.begin(), shape.end(), 1, std::multiplies{});
            std::vector<int8_t> h_out(elems);
            std::vector<DataType> h_input(elems);
            std::vector<DataType> h_scales(shape[0] * shape[2]);

            check_cuda_error(hipMemcpy(h_input.data(), inputs, elems * sizeof(DataType), hipMemcpyDeviceToHost));

            cutlass_kernels::symmetric_quantize(h_out.data(), h_scales.data(), h_input.data(), shape, quant_type, true);

            check_cuda_error(hipMemcpy(
                outputs, h_out.data(), elems * sizeof(int8_t) / WEIGHT_ELEM_PER_BYTE, hipMemcpyHostToDevice));
            check_cuda_error(
                hipMemcpy(scales, h_scales.data(), h_scales.size() * sizeof(DataType), hipMemcpyHostToDevice));
        }
    }

    constexpr static float getFP8Scalar(float in)
    {
        return FP8_MAX / in;
    }

    void initFP8Scales(float max_input)
    {
        check_cuda_error(hipStreamSynchronize(mStream->get()));

        // Add shift to the max because we add an adjustment for each expert so they get different results.
        float max_shift = expertShift(mNumExperts - 1, mNumExperts);
        float maxW1 = max_shift + (mIsGated ? std::max(mExpertWDiag1, mExpertWDiagGated) : mExpertWDiag1);
        float maxW2 = max_shift + mExpertWDiag2;
        float scaleW1 = getFP8Scalar(maxW1);
        float scaleW2 = getFP8Scalar(maxW2);
        mFP8WeightScalar1 = scaleW1;
        mFP8WeightScalar2 = scaleW2;

        float scaleAct1 = getFP8Scalar(max_input);

        float maxFC1Output = calcMLPVal(max_input, mNumExperts - 1) / maxW2;
        float scaleAct2 = getFP8Scalar(maxFC1Output);

        ASSERT_NE(mExpertFP8Scale1, nullptr);
        ASSERT_NE(mExpertFP8Scale2, nullptr);
        ASSERT_NE(mExpertFP8Scale3, nullptr);
        // Dequant values for each expert are 1/(w_i*a_i) calculated above
        std::fill_n(mExpertFP8Scale1, mNumExperts, 1.f / (scaleW1 * scaleAct1));
        std::fill_n(mExpertFP8Scale3, mNumExperts, 1.f / (scaleW2 * scaleAct2));
        *mExpertFP8Scale2 = scaleAct2;

        check_cuda_error(hipStreamSynchronize(mStream->get()));
    }

    void resetOutBuffers()
    {
        auto stream = mStream->get();

        if (mTpExpertScratch)
            check_cuda_error(hipMemsetAsync(mTpExpertScratch, 0x0, mTpExpertScratchSize, stream));
        check_cuda_error(hipMemsetAsync(mFinalOutput, 0x0, mTotalTokens * mHiddenSize * sizeof(DataType), stream));
        check_cuda_error(hipMemsetAsync(mSourceToExpandedMap, 0x0, sizeof(int) * mTotalTokens * mK, stream));
        check_cuda_error(hipMemsetAsync(mSelectedExpert, 0x0, sizeof(int) * mTotalTokens * mK, stream));
        check_cuda_error(hipMemsetAsync(mScaleProbs, 0x0, sizeof(float) * mTotalTokens * mK, stream));

        check_cuda_error(hipStreamSynchronize(stream));
    }

    void resizeRouterInputs(
        std::vector<std::vector<float>>& h_router_results, int64_t num_experts, int64_t num_tokens_per_seq)
    {
        for (int64_t i = 0; i < h_router_results.size(); i++)
        {
            auto& seq_routing = h_router_results[i];
            int64_t num_tokens = num_tokens_per_seq;
            auto hardcoded_experts = seq_routing.size() / num_tokens;
            ASSERT_EQ(seq_routing.size(), hardcoded_experts * num_tokens);
            if (num_experts > hardcoded_experts)
            {
                auto pos = seq_routing.begin() + hardcoded_experts;
                for (int64_t i = 0; i < num_tokens; i++, pos += num_experts)
                {
                    pos = seq_routing.insert(pos, num_experts - hardcoded_experts, 0);
                }
            }
            ASSERT_EQ(seq_routing.size(), num_experts * num_tokens);
        }
    }

    template <class T>
    auto populateTokens(std::vector<T>& hidden_states)
    {
        if constexpr (std::is_same_v<T, SafeFP8>)
        {
            std::vector<OutputType> internal_states(hidden_states.size());
            populateTokens(internal_states);

            mMaxInput = *std::max_element(internal_states.begin(), internal_states.end());
            float scalar = getFP8Scalar(mMaxInput);
            std::transform(internal_states.begin(), internal_states.end(), hidden_states.begin(),
                [scalar](OutputType in) -> T { return static_cast<T>((float) in * scalar); });
            // Do the reverse transformation since we only have so much precision and this is a pretty broad range
            std::transform(hidden_states.begin(), hidden_states.end(), internal_states.begin(),
                [scalar](T in) -> OutputType { return static_cast<OutputType>(((float) in) / scalar); });
            return internal_states;
        }
        else
        {
            std::vector<float> base(hidden_states.size());
            std::iota(base.begin(), base.end(), 0.0f);
            // Lambda subtracts a small value so we have some < 0 to test the activation for negatives
            std::transform(base.begin(), base.end(), hidden_states.begin(),
                [l = hidden_states.size()](auto a) { return T(a / l) - T(0.01f); });
            return hidden_states;
        }
    }

    void runMoEPermute(std::vector<std::vector<DataType>> h_hidden_states,
        std::vector<std::vector<float>> h_router_results, int64_t hidden_size, int64_t num_experts, int64_t k,
        std::vector<uint8_t> finished = {}, MOEParallelismConfig parallelism_config = {})
    {
        initBuffersPermute(std::move(h_hidden_states), std::move(h_router_results), hidden_size, num_experts, k,
            finished, parallelism_config);
        runMoEPermute(parallelism_config);
    }

    auto getWeights(MOEParallelismConfig parallelism_config)
    {
        void* scale_1 = FP8 ? (void*) mExpertFP8Scale1 : (void*) mExpertIntScale1;
        void* scale_2 = FP8 ? (void*) mExpertFP8Scale2 : (void*) mExpertIntScale2;
        void* scale_3 = FP8 ? mExpertFP8Scale3 : nullptr;

        if (parallelism_config.tp_size > 1)
        {
            int const tp_size = parallelism_config.tp_size;
            int const tp_rank = parallelism_config.tp_rank;

            size_t const matrix_size = mHiddenSize * mInterSize / tp_size;
            size_t const gated_matrix_size = mHiddenSize * mInterSize * mGatedMultiplier / tp_size;
            size_t const row_size_inter = mInterSize / tp_size;
            size_t const gated_row_size_inter = mInterSize * mGatedMultiplier / tp_size;
            size_t const gated_bias_size = mUseBias ? gated_row_size_inter : 0;

            auto* weight_1 = reinterpret_cast<WeightStorage*>(mTpExpertScratch);
            auto* weight_2 = weight_1 + mNumExperts * gated_matrix_size;
            auto* bias_1 = reinterpret_cast<DataType*>(weight_2 + mNumExperts * matrix_size);
            auto* int_scale_1 = bias_1 + mNumExperts * gated_bias_size;

            // 2D memcpy just the slices we care about
            // TODO Re-quantize here with matrices divided
            size_t const row_size_1 = matrix_size * sizeof(WeightStorage) / WEIGHT_ELEM_PER_BYTE;
            check_cuda_error(hipMemcpy2DAsync(weight_1, row_size_1, (uint8_t*) mExpertWeight1 + row_size_1 * tp_rank,
                row_size_1 * tp_size, row_size_1, mNumExperts * mGatedMultiplier, hipMemcpyDeviceToDevice,
                mStream->get()));

            size_t const row_size_2 = row_size_inter * sizeof(WeightStorage) / WEIGHT_ELEM_PER_BYTE;
            check_cuda_error(hipMemcpy2DAsync(weight_2, row_size_2, (uint8_t*) mExpertWeight2 + row_size_2 * tp_rank,
                row_size_2 * tp_size, row_size_2, mNumExperts * mHiddenSize, hipMemcpyDeviceToDevice, mStream->get()));

            if (mUseBias)
            {
                size_t const row_size_bias = row_size_inter * sizeof(DataType);
                check_cuda_error(hipMemcpy2DAsync(bias_1, row_size_bias,
                    (uint8_t*) mExpertBias1 + row_size_bias * tp_rank, row_size_bias * tp_size, row_size_bias,
                    mNumExperts * mGatedMultiplier, hipMemcpyDeviceToDevice, mStream->get()));
            }

            if constexpr (INT_QUANT)
            {
                scale_2 = mExpertIntScale2;
                size_t const row_size_scale = row_size_inter * sizeof(DataType);
                check_cuda_error(hipMemcpy2DAsync(scale_1, row_size_scale,
                    (uint8_t*) mExpertIntScale1 + row_size_scale * tp_rank, row_size_scale * tp_size, row_size_scale,
                    mNumExperts * mGatedMultiplier, hipMemcpyDeviceToDevice, mStream->get()));
            }

            bias_1 = mUseBias ? bias_1 : nullptr;
            return std::tuple{weight_1, weight_2, bias_1, mExpertBias2, scale_1, scale_2, scale_3};
        }
        else if (parallelism_config.ep_size > 1)
        {
            size_t const gated_inter = mInterSize * mGatedMultiplier;
            size_t const experts_per_node = mNumExperts / parallelism_config.ep_size;
            size_t const weight_matrix_size = mHiddenSize * mInterSize * experts_per_node / WEIGHT_ELEM_PER_BYTE;
            size_t const bias_fc1_size = gated_inter * experts_per_node;
            size_t const bias_fc2_size = mHiddenSize * experts_per_node;
            size_t const scale1_size = gated_inter * experts_per_node;
            size_t const scale2_size = mHiddenSize * experts_per_node;
            auto* weight1_ptr = mExpertWeight1 + weight_matrix_size * mGatedMultiplier * parallelism_config.ep_rank;
            auto* weight2_ptr = mExpertWeight2 + weight_matrix_size * parallelism_config.ep_rank;
            auto* bias1_ptr = mUseBias ? mExpertBias1 + bias_fc1_size * parallelism_config.ep_rank : nullptr;
            auto* bias2_ptr = mUseBias ? mExpertBias2 + bias_fc2_size * parallelism_config.ep_rank : nullptr;

            if (INT_QUANT)
            {
                scale_1 = mExpertIntScale1 + scale1_size * parallelism_config.ep_rank;
                scale_2 = mExpertIntScale2 + scale2_size * parallelism_config.ep_rank;
            }
            if constexpr (FP8)
            {
                scale_1 = mExpertFP8Scale1 + experts_per_node * parallelism_config.ep_rank;
                scale_3 = mExpertFP8Scale3 + experts_per_node * parallelism_config.ep_rank;
            }

            return std::tuple{weight1_ptr, weight2_ptr, bias1_ptr, bias2_ptr, scale_1, scale_2, scale_3};
        }

        return std::tuple{mExpertWeight1, mExpertWeight2, mExpertBias1, mExpertBias2, scale_1, scale_2, scale_3};
    }

    void runMoEPermute(MOEParallelismConfig parallelism_config)
    {
        // Clear the buffers to blank so we can assume zero if not written
        resetOutBuffers();

        auto const [weight1_ptr, weight2_ptr, bias1_ptr, bias2_ptr, scale1_ptr, scale2_ptr, scale3_ptr]
            = getWeights(parallelism_config);

        auto stream = mStream->get();
        auto tactic = mSelectedConfig;
        if (!tactic)
        {
            int sm = getSMVersion();
            bool is_sm90 = sm >= 90 && !INT_QUANT;
            auto tactics = mMoERunner.getTactics();
            auto it = std::find_if(tactics.begin(), tactics.end(), [is_sm90](auto& c) { return c.is_sm90 == is_sm90; });
            if (it == tactics.end())
            {
                // Fall back to any tactic
                std::cout << "WARNING: Could not find config for sm version " << sm << std::endl;
                tactic = tactics[0];
            }
            else
            {
                tactic = *it;
            }
        }

        QuantParams quant_params;
        if constexpr (INT_QUANT)
        {
            quant_params = QuantParams::Int(scale1_ptr, scale2_ptr);
        }
        else
        {
            quant_params = QuantParams::FP8(static_cast<float const*>(scale1_ptr),
                static_cast<float const*>(scale2_ptr), static_cast<float const*>(scale3_ptr));
        }

        mMoERunner.setTactic(tactic);
        mMoERunner.runMoe(mInputTensor, mInputProbabilities, weight1_ptr, bias1_ptr, mActType, weight2_ptr, bias2_ptr,
            quant_params, mTotalTokens, mHiddenSize, mInterSize / parallelism_config.tp_size, mNumExperts, mK,
            mWorkspace, mFinalOutput, mFinished, mActiveRows, mScaleProbs, mSourceToExpandedMap, mSelectedExpert,
            parallelism_config, mNormMode, stream);

        check_cuda_error(hipStreamSynchronize(stream));
    }

    template <class T>
    std::vector<T> getDataFromDevice(T const* in, size_t length)
    {
        std::vector<T> data(length);

        auto const stream = mStream->get();
        check_cuda_error(hipMemcpyAsync(data.data(), in, length * sizeof(T), hipMemcpyDeviceToHost, stream));
        check_cuda_error(hipStreamSynchronize(mStream->get()));

        return data;
    }

    auto maskSelectedExpertsForTP(std::vector<int> const& vector, int tp_size, int tp_rank)
    {
        std::vector<int> result;
        int num_experts_per_node = mNumExperts / tp_size;
        std::transform(vector.begin(), vector.end(), std::back_inserter(result),
            [=](int entry)
            {
                if (entry >= num_experts_per_node * tp_rank && entry < num_experts_per_node * (tp_rank + 1))
                    return entry;
                return (int) mNumExperts;
            });
        return result;
    }

    void debugPrint()
    {
#define PRINT_CAST(array, size, cast)                                                                                  \
    do                                                                                                                 \
        if (array)                                                                                                     \
        {                                                                                                              \
            auto data = getDataFromDevice(array, size);                                                                \
            std::cout << #array << ": ";                                                                               \
            for (auto v : data)                                                                                        \
            {                                                                                                          \
                if (cast(v))                                                                                           \
                    std::cout << cast(v) << ", ";                                                                      \
                else                                                                                                   \
                    std::cout << "., ";                                                                                \
            }                                                                                                          \
            std::cout << std::endl;                                                                                    \
        }                                                                                                              \
    while (0)
#define PRINT(array, size) PRINT_CAST(array, size, )

        using WeightPrintType = std::conditional_t<INT_QUANT, uint8_t, WeightStorage>;
        PRINT_CAST((WeightPrintType*) mExpertWeight1,
            mNumExperts * mHiddenSize * mInterSize * mGatedMultiplier / WEIGHT_ELEM_PER_BYTE, float);
        PRINT_CAST(
            (WeightPrintType*) mExpertWeight2, mNumExperts * mHiddenSize * mInterSize / WEIGHT_ELEM_PER_BYTE, float);
        // PRINT_CAST(mRawExpertWeight1, mNumExperts * mHiddenSize * mInterSize * mGatedMultiplier, float);
        // PRINT_CAST(mRawExpertWeight2, mNumExperts * mHiddenSize * mInterSize, float);
        PRINT_CAST(mExpertBias1, mNumExperts * mInterSize * mGatedMultiplier, float);
        PRINT_CAST(mExpertBias2, mNumExperts * mHiddenSize, float);
        PRINT_CAST(mExpertIntScale1, mNumExperts * mInterSize * mGatedMultiplier, float);
        PRINT_CAST(mExpertIntScale2, mNumExperts * mHiddenSize, float);
        PRINT(mFinalOutput, mTotalTokens * mHiddenSize);
        PRINT_CAST((uint8_t*) mFinished, mTotalTokens, (int) );
        PRINT(mInputProbabilities, mTotalTokens * mNumExperts);
        PRINT(mScaleProbs, mTotalTokens * mK);
        PRINT(mInputProbabilities, mTotalTokens * mNumExperts);
        PRINT_CAST(mInputTensor, mTotalTokens * mHiddenSize, float);
        PRINT(mSourceToExpandedMap, mTotalTokens * mK);
        PRINT(mSelectedExpert, mTotalTokens * mK);

#undef PRINT_CAST
#undef PRINT
    }

    template <class T>
    T actfn(T in)
    {
        if (mActType == tensorrt_llm::ActivationType::Identity)
            return in;
        if (mActType == tensorrt_llm::ActivationType::Relu)
            return std::max(in, T(0.0f));
        if (mActType == tensorrt_llm::ActivationType::Gelu || mActType == tensorrt_llm::ActivationType::Geglu)
            return (std::erf(float(in) * float(sqrt(0.5))) + 1) * 0.5f * float(in);
        assert(false);
        return in;
    }

    float calcMLPVal(float input, int expert_id, bool final_bias = false)
    {
        if (expert_id >= mNumExperts)
            return 0;

        float expert_shift = expertShift(expert_id, mNumExperts);
        float w1_bias = mUseBias ? expert_id : 0.f;
        float activated = 0;
        if (mIsGated)
        {
            float scalar = mExpertWDiag1 + expert_shift;
            float fc1 = (float) input * scalar + (float) w1_bias;

            float gated_scalar = mExpertWDiagGated + expert_shift;
            float gated_bias = mUseBias ? (float) w1_bias + 1.f : 0.f;
            float gate = (float) input * gated_scalar + gated_bias;

            activated = fc1 * actfn(gate);
        }
        else
        {
            float scalar = mExpertWDiag1 + expert_shift;
            float fc1 = input * scalar + w1_bias;
            activated = actfn(fc1);
        }

        EXPECT_TRUE(mUseBias || !final_bias);
        float result = activated * (mExpertWDiag2 + expert_shift) + (float) (final_bias ? expert_id : 0);
        return result;
    }

    float calcMLPValWithFinalBias(float input, int expert_id)
    {
        return calcMLPVal(input, expert_id, mUseBias);
    }

    // NOTE This is a useful function for debugging routing failures. But you need to know the exact offset of
    //   this info in the workspace so having a test depend on something so internal is suboptimal
    //
    // void comparePermuted(const std::vector<int>& expected_experts, const std::vector<int>& expected_permutation,
    //     const std::vector<DataType>& input_data)
    //{
    //     auto states = getDataFromDevice(magic incantation into workspace, mTotalTokens * mK * mHiddenSize);
    //
    //    // Loop for the number of times each token is duplicated
    //    for (int k_idx = 0; k_idx < mK; k_idx++)
    //    {
    //        for (int64_t token_id = 0; token_id < mTotalTokens; token_id++)
    //        {
    //            // Permutation has the position of the first copy of all token,
    //            // followed by the position of the second copy of all tokens etc.
    //            const int64_t permuted_position = expected_permutation[k_idx * mTotalTokens + token_id];
    //
    //            // Expected experts has all the selected experts for token one,
    //            // followed by all the selected experts for token two etc.
    //            const int64_t expert_id = expected_experts[token_id * mK + k_idx];
    //
    //            // Compare the copied tokens with the projection applied
    //            for (int64_t hidden_id = 0; hidden_id < mHiddenSize; hidden_id++)
    //            {
    //                auto ref = calcMLPVal(input_data[token_id * mHiddenSize + hidden_id], expert_id);
    //                auto actual = states[permuted_position * mHiddenSize + hidden_id];
    //                ASSERT_NEAR(ref, actual, getTolerance(ref))
    //                    << "Incorrect value at position: mK: " << k_idx << ", token: " << token_id
    //                    << ", permuted dest: " << permuted_position << ", expert id: " << expert_id
    //                    << ", hidden id: " << hidden_id;
    //            }
    //        }
    //    }
    //}

    std::vector<float> softmax(std::vector<float> const& expected_probs)
    {
        std::vector<float> softmax;
        // All values we test are 0-1 so we can skip the normalization step
        std::transform(expected_probs.begin(), expected_probs.end(), std::back_inserter(softmax),
            [&](float const in) -> float
            {
                auto res = exp(in);
                return res;
            });

        for (int64_t token = 0; token < mTotalTokens; token++)
        {
            auto start = softmax.begin() + token * mNumExperts;
            auto end = start + mNumExperts;
            auto sum = std::accumulate(start, end, 0.f);
            std::transform(start, end, start, [=](auto in) { return in / sum; });
        }

        return softmax;
    }

    void compareSoftmax(std::vector<int> const& expected_experts, std::vector<float> const& expected_probs,
        std::vector<float> scale_probs = {})
    {
        if (scale_probs.empty())
            scale_probs = getDataFromDevice(mScaleProbs, mTotalTokens * mK);
        auto softmax_probs = softmax(expected_probs);
        for (int64_t token_id = 0; token_id < mTotalTokens; token_id++)
        {
            for (int k_idx = 0; k_idx < mK; k_idx++)
            {
                int selected_expert = expected_experts[token_id * mK + k_idx];
                if (selected_expert < mNumExperts) // Ignore 'finished' values
                {
                    ASSERT_NEAR(softmax_probs[token_id * mNumExperts + selected_expert],
                        scale_probs[token_id * mK + k_idx], getTolerance())
                        << "Scales mismatched for token: " << token_id << " k: " << k_idx
                        << " selected_expert: " << selected_expert;
                }
            }
        }
    }

    void renormScales(float* probs, int const* experts)
    {
        if (mNormMode == MOEExpertScaleNormalizationMode::NONE)
            return;
        float sum = 0;
        for (int k_idx = 0; k_idx < mK; k_idx++)
        {
            sum += probs[experts[k_idx]];
        }
        float norm_factor = 1.0f / sum;
        for (int k_idx = 0; k_idx < mK; k_idx++)
        {
            probs[experts[k_idx]] *= norm_factor;
        }
    }

    void compareFinal(std::vector<int> const& expected_experts, std::vector<float> const& expected_probs,
        std::vector<OutputType> const& input_data, std::vector<OutputType> final_results = {})
    {
        if (final_results.empty())
            final_results = getDataFromDevice(mFinalOutput, mTotalTokens * mHiddenSize);

        auto softmax_probs = softmax(expected_probs);
        for (int64_t token_id = 0; token_id < mTotalTokens; token_id++)
        {
            renormScales(&softmax_probs[token_id * mNumExperts], &expected_experts[token_id * mK]);

            for (int64_t hidden_id = 0; hidden_id < mHiddenSize; hidden_id++)
            {
                float sum = 0.0f;
                // Loop for the number of times each token is duplicated
                for (int k_idx = 0; k_idx < mK; k_idx++)
                {
                    int selected_expert = expected_experts[token_id * mK + k_idx];
                    sum += float(calcMLPValWithFinalBias(
                               static_cast<float>(input_data[token_id * mHiddenSize + hidden_id]), selected_expert))
                        * softmax_probs[token_id * mNumExperts + selected_expert];
                }

                ASSERT_NEAR(OutputType{sum}, final_results[token_id * mHiddenSize + hidden_id], getTolerance(sum))
                    << "Incorrect final value at position: " << token_id * mHiddenSize + hidden_id;
            }
        }
    }

    void BasicPermuteTest(int k = 1, int64_t hidden_size = DEFAULT_HIDDEN_SIZE);

    std::vector<int> calcPermuteMapExpertParallel(std::vector<int> const& expected_experts);
    void ExpertParallelTest(int k = 1);

    void TensorParallelTest(int k = 1);
};

template <class WeightParams>
using LargeMixtureOfExpertsTest = MixtureOfExpertsTest<WeightParams>;

template <class DataType_, class WeightType_ = DataType_, class OutputType_ = DataType_>
struct WeightParams
{
    using DataType = DataType_;
    using WeightType = WeightType_;
    using OutputType = OutputType_;
};

// TODO Fix int quantized
using Types = ::testing::Types<
#ifdef ENABLE_BF16
    WeightParams<__hip_bfloat16>,
#endif
#ifdef ENABLE_FP8
    WeightParams<SafeFP8, SafeFP8, half>,
#endif
    WeightParams<half>, WeightParams<float>

    //, WeightParams<half, uint8_t>, WeightParams<half, cutlass::uint4b_t>

    >;
TYPED_TEST_SUITE(MixtureOfExpertsTest, Types);
// Have a separate test with only one data type because this test is long
TYPED_TEST_SUITE(LargeMixtureOfExpertsTest, ::testing::Types<WeightParams<half>>);

template <class TypeParam_>
BufferManager::CudaStreamPtr MixtureOfExpertsTest<TypeParam_>::mStream{};
template <class TypeParam_>
std::unique_ptr<BufferManager> MixtureOfExpertsTest<TypeParam_>::mBufferManager{};
template <class TypeParam_>
int MixtureOfExpertsTest<TypeParam_>::mDeviceCount{};

template <class TypeParam_>
void MixtureOfExpertsTest<TypeParam_>::BasicPermuteTest(int k, int64_t hidden_size)
{
    if constexpr (FP8)
    {
        // TODO Remove this when bias + FP8 is supported
        mUseBias = false;
    }

    int64_t num_experts = 4;
    int64_t num_tokens = 3;

    std::vector<DataType> hidden_states(hidden_size * num_tokens);
    auto raw_unquant_input = populateTokens(hidden_states);

    std::vector<float> probs = {
        0.5, 0.1, 0.25, 0.15,   //
        0.03, 0.2, 0.07, 0.7,   //
        0.25, 0.21, 0.35, 0.19, //
    };

    std::vector<std::vector<DataType>> hidden_input = {hidden_states};
    std::vector<std::vector<float>> router_input = {probs};
    resizeRouterInputs(router_input, num_experts, num_tokens);

    runMoEPermute(hidden_input, router_input, hidden_size, num_experts, k);

    std::vector<int> expected_experts{0, 3, 2};
    if (k == 2)
        expected_experts = {0, 2, 3, 1, 2, 0};
    else if (k == 3)
        expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};

    auto selected_expert = getDataFromDevice(mSelectedExpert, num_tokens * k);
    EXPECT_EQ(selected_expert, expected_experts);

    auto proj_map = getDataFromDevice(mSourceToExpandedMap, num_tokens * k);
    // This is the final position of:
    // Token 1 Expert 1, T2E1, T3E1, T1E2, T2E2, T3E2
    std::vector<int> permute_map{0, 2, 1};
    if (k == 2)
        permute_map = {0, 5, 4, 3, 2, 1};
    if (k == 3)
        permute_map = {0, 8, 6, 4, 2, 1, 7, 5, 3};
    ASSERT_EQ(permute_map, proj_map);
    compareSoftmax(selected_expert, router_input[0]);
    compareFinal(selected_expert, router_input[0], raw_unquant_input);
}

TYPED_TEST(MixtureOfExpertsTest, Permute)
{
    this->BasicPermuteTest();
}

TYPED_TEST(MixtureOfExpertsTest, PermuteK2)
{
    this->BasicPermuteTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteK3)
{
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteNoBias)
{
    this->mUseBias = false;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteRenormalization)
{
    this->mNormMode = tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteGeglu)
{
    this->mActType = tensorrt_llm::ActivationType::Geglu;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, Finished)
{
    if (this->FP8)
    {
        // TODO Remove this when bias + FP8 is supported
        this->mUseBias = false;
    }

    using DataType = typename TypeParam::DataType;
    int64_t hidden_size = this->DEFAULT_HIDDEN_SIZE;
    int64_t num_experts = 4;
    int64_t num_tokens = 3;
    int64_t k = 2;

    std::vector<DataType> hidden_states(hidden_size * num_tokens);
    auto raw_unquant_input = this->populateTokens(hidden_states);

    std::vector<float> probs = {
        0.5, 0.1, 0.25, 0.15, //
        0.05, 0.2, 0.05, 0.7, //
        0.25, 0.2, 0.35, 0.2, //
    };

    this->runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k, {0, 0, 1});

    auto selected_expert = this->getDataFromDevice(this->mSelectedExpert, num_tokens * k);
    // Token 1
    EXPECT_EQ(selected_expert[0], 0);
    EXPECT_EQ(selected_expert[1], 2);
    // Token 2
    EXPECT_EQ(selected_expert[2], 3);
    EXPECT_EQ(selected_expert[3], 1);
    // Token 3
    EXPECT_EQ(selected_expert[4], num_experts); // One past the end
    EXPECT_EQ(selected_expert[5], num_experts);

    auto proj_map = this->getDataFromDevice(this->mSourceToExpandedMap, num_tokens * k);
    // This is the final position of:
    // Token 1 Expert 1, T2E1, T3E1, T1E2, T2E2, T3E3
    std::vector<int> permute_map{0, 3, 4, 2, 1, 5};
    ASSERT_EQ(permute_map, proj_map);
    this->compareSoftmax(selected_expert, probs);
    this->compareFinal(selected_expert, probs, raw_unquant_input);
}

template <class TypeParam_>
std::vector<int> MixtureOfExpertsTest<TypeParam_>::calcPermuteMapExpertParallel(
    std::vector<int> const& expected_experts)
{
    std::vector<int> map(expected_experts.size());
    auto getInterleavedIndex = [this](int i) { return (i % mK) * mTotalTokens + i / mK; };
    int map_idx = 0;
    for (int expert = 0; expert <= mNumExperts; expert++)
    {
        for (int i = 0; i < map.size(); i++)
        {
            if (expected_experts[i] == expert)
                map[getInterleavedIndex(i)] = map_idx++;
        }
    }

    return map;
}

template <class TypeParam_>
void MixtureOfExpertsTest<TypeParam_>::ExpertParallelTest(int k)
{
    if (FP8)
    {
        // TODO Remove this when bias + FP8 is supported
        mUseBias = false;
    }

    int64_t hidden_size = DEFAULT_HIDDEN_SIZE;
    int64_t parallelism = 2;
    int64_t num_experts = 4;
    int64_t num_tokens = 3;

    std::vector<DataType> hidden_states(hidden_size * num_tokens);
    auto raw_unquant_input = populateTokens(hidden_states);

    std::vector<float> probs = {
        0.5, 0.1, 0.25, 0.15,   //
        0.03, 0.2, 0.07, 0.7,   //
        0.25, 0.21, 0.35, 0.19, //
    };

    std::vector<int> expected_experts{0, 3, 2};
    if (k == 2)
        expected_experts = {0, 2, 3, 1, 2, 0};
    else if (k == 3)
        expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};
    std::vector<OutputType> results(hidden_states.size(), 0);
    for (int i = 0; i < parallelism; i++)
    {
        if (i == 0)
        {
            // Only need to init the inputs on the first iteration
            runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k, {},
                MOEParallelismConfig::ExpertParallelism(parallelism, i));
        }
        else
        {
            runMoEPermute(MOEParallelismConfig::ExpertParallelism(parallelism, i));
        }

        auto selected_expert = getDataFromDevice(mSelectedExpert, num_tokens * k);
        // Experts should only be selected when we are on the right node
        // Note the index is [0,num_experts_per_node), so we offset the experts by the start for this node
        int const start_expert = i * (mNumExperts / parallelism);
        std::transform(selected_expert.begin(), selected_expert.end(), selected_expert.begin(),
            [&](int val) { return val == mNumExperts ? mNumExperts : val + start_expert; });
        auto masked_expected_experts = maskSelectedExpertsForTP(expected_experts, parallelism, i);
        ASSERT_EQ(selected_expert, masked_expected_experts);

        auto proj_map = getDataFromDevice(mSourceToExpandedMap, num_tokens * k);
        auto permute_map = calcPermuteMapExpertParallel(masked_expected_experts);
        ASSERT_EQ(permute_map, proj_map) << "Iteration " << i;
        compareSoftmax(expected_experts, probs);

        // Do the final reduce
        auto iter_results = getDataFromDevice(mFinalOutput, num_tokens * hidden_size);
        std::transform(iter_results.cbegin(), iter_results.cend(), results.cbegin(), results.begin(), std::plus<>{});
    }

    compareFinal(expected_experts, probs, raw_unquant_input, results);
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallel)
{
    this->ExpertParallelTest();
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallelK2)
{
    this->ExpertParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallelNoBias)
{
    this->mUseBias = false;
    this->ExpertParallelTest();
    this->ExpertParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallelRenorm)
{
    this->mNormMode = MOEExpertScaleNormalizationMode::RENORMALIZE;
    this->ExpertParallelTest();
    this->ExpertParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallelGeglu)
{
    this->mActType = tensorrt_llm::ActivationType::Geglu;
    this->ExpertParallelTest();
    this->ExpertParallelTest(2);
}

template <class TypeParam_>
void MixtureOfExpertsTest<TypeParam_>::TensorParallelTest(int k)
{
    if (FP8)
    {
        // TODO Remove this when bias + FP8 is supported
        mUseBias = false;
    }

    int64_t hidden_size = DEFAULT_HIDDEN_SIZE;
    int64_t parallelism = 8;
    int64_t num_experts = 4;
    int64_t num_tokens = 3;

    std::vector<DataType> hidden_states(hidden_size * num_tokens);
    auto raw_unquant_input = populateTokens(hidden_states);

    std::vector<float> probs = {
        0.5, 0.1, 0.25, 0.15,   //
        0.03, 0.2, 0.07, 0.7,   //
        0.25, 0.21, 0.35, 0.19, //
    };

    std::vector<int> expected_experts{0, 3, 2};
    if (k == 2)
        expected_experts = {0, 2, 3, 1, 2, 0};
    else if (k == 3)
        expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};
    std::vector<OutputType> results(hidden_states.size(), 0);
    for (int i = 0; i < parallelism; i++)
    {
        if (i == 0)
        {
            // Only need to init the inputs on the first iteration
            runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k, {},
                MOEParallelismConfig::TensorParallelism(parallelism, i));
        }
        else
        {
            runMoEPermute(MOEParallelismConfig::TensorParallelism(parallelism, i));
        }

        auto selected_expert = getDataFromDevice(mSelectedExpert, num_tokens * k);
        EXPECT_EQ(selected_expert, expected_experts);

        auto proj_map = getDataFromDevice(mSourceToExpandedMap, num_tokens * k);
        std::vector<int> permute_map{0, 2, 1};
        if (k == 2)
            permute_map = {0, 5, 4, 3, 2, 1};
        if (k == 3)
            permute_map = {0, 8, 6, 4, 2, 1, 7, 5, 3};

        ASSERT_EQ(permute_map, proj_map) << "Iteration " << i;

        // Do the final reduce
        auto iter_results = getDataFromDevice(mFinalOutput, num_tokens * hidden_size);
        std::transform(iter_results.cbegin(), iter_results.cend(), results.cbegin(), results.begin(), std::plus<>{});
    }

    compareFinal(expected_experts, probs, raw_unquant_input, results);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallel)
{
    this->TensorParallelTest();
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelK2)
{
    this->TensorParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelK3)
{
    this->TensorParallelTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelNoBias)
{
    this->mUseBias = false;
    this->TensorParallelTest();
    this->TensorParallelTest(2);
    this->TensorParallelTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelRenorm)
{
    this->mNormMode = MOEExpertScaleNormalizationMode::RENORMALIZE;
    this->TensorParallelTest();
    this->TensorParallelTest(2);
    this->TensorParallelTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelGeglu)
{
    this->mActType = tensorrt_llm::ActivationType::Geglu;
    this->TensorParallelTest();
    this->TensorParallelTest(2);
    this->TensorParallelTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, ConfigSweep)
{
    auto configs = this->mMoERunner.getTactics();
    for (auto conf : configs)
    {
        using namespace tensorrt_llm::cutlass_extensions;
        std::stringstream tactic;
        tactic << "Failed " << (conf.is_sm90 ? "SM90+" : "<SM90") << " tactic with tile shape ";
        if (conf.tile_config_sm90 != CutlassTileConfigSM90::ChooseWithHeuristic)
        {
            tactic << (int) conf.tile_config_sm90 << " and cluster shape " << (int) conf.cluster_shape
                   << " mainloop sched " << (int) conf.mainloop_schedule << " epi sched "
                   << (int) conf.epilogue_schedule;
        }
        else if (conf.tile_config != CutlassTileConfig::ChooseWithHeuristic)
        {
            tactic << (int) conf.tile_config << " and stages " << (int) conf.stages << " split k "
                   << (int) conf.split_k_factor;
        }
        else
        {
            FAIL() << "Uninitialised tactic encountered";
        }

        EXPECT_NO_THROW({
            this->mSelectedConfig = conf;
            this->BasicPermuteTest();
            if (::testing::Test::HasFailure())
                throw std::runtime_error("Test Failed");
        }) << tactic.str();
    }
}

TYPED_TEST(LargeMixtureOfExpertsTest, PermuteVeryLargeExperts)
{
    // Chosen so that hidden_size * inter_size * num_experts >> 2^32, but we can still fit in 80GB for `half`
    // Uses a non-power of two so any integer overflow will have bad alignment
    int64_t hidden_size = 31 * 1024;
    ASSERT_GT(hidden_size * hidden_size * 4, (int64_t) std::numeric_limits<int>::max() + 1ull);

    int64_t k = 2; // Use k=2 so all experts get a value
    // 3 tokens 4 experts are the defaults for BasicPermuteTest
    if (!this->checkSufficientTestMemory(3, hidden_size, 4, k))
    {
        GTEST_SKIP() << "Insufficient free memory for test";
    }

    this->BasicPermuteTest(k, hidden_size); // 4 x 32k x 128K experts
}

TYPED_TEST(LargeMixtureOfExpertsTest, PermuteVeryLongSequence)
{
    this->mUseBias = !this->FP8;

    using DataType = typename TypeParam::DataType;
    // Sequence * hidden size > INT32_MAX
    int64_t hidden_size = 2048ll;
    int64_t num_experts = 4;
    int64_t k = 1;
    int64_t num_tokens = 1024ll * 1024ll + 1ll;
    ASSERT_GT(hidden_size * num_tokens, (uint64_t) std::numeric_limits<int>::max() + 1ull);

    if (!this->checkSufficientTestMemory(num_tokens, hidden_size, num_experts, k))
    {
        GTEST_SKIP() << "Insufficient free memory for test";
    }

    std::vector<DataType> hidden_states(hidden_size * num_tokens);
    this->mMaxInput = 1.f; // Any arbitrary non-zero value

    // All tokens to expert 0
    float const token_probs[] = {1.f, 0.5f, 0.f, 0.f};
    std::vector<float> probs;
    probs.reserve(num_tokens * num_experts);
    for (size_t i = 0; i < num_tokens; i++)
    {
        probs.insert(probs.cend(), std::begin(token_probs), std::end(token_probs));
    }

    this->runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k);

    // Just look at the first few tokens
    this->mTotalTokens = 10;

    probs.resize(num_experts * this->mTotalTokens);
    hidden_states.resize(hidden_size * this->mTotalTokens);

    auto selected_expert = this->getDataFromDevice(this->mSelectedExpert, k * this->mTotalTokens);
    // All tokens should go to expert 0
    for (auto& item : selected_expert)
    {
        ASSERT_EQ(item, 0);
    }

    this->compareSoftmax(selected_expert, probs);
    // Create a default vector for the reference outputs of the correct type for FP8
    std::vector<typename TypeParam::OutputType> unquant_states(this->mTotalTokens * hidden_size);
    this->compareFinal(selected_expert, probs, unquant_states);
}
