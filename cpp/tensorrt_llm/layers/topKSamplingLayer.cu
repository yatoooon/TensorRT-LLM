#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include "tensorrt_llm/kernels/samplingTopKKernels.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"
#include "tensorrt_llm/layers/defaultDecodingParams.h"
#include "tensorrt_llm/layers/layerUtils.h"
#include "tensorrt_llm/layers/topKSamplingLayer.h"
#include "tensorrt_llm/runtime/iTensor.h"

#include <algorithm>
#include <float.h>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm
{
namespace layers
{

template <int32_t TOP_K_MAX>
__global__ void setupTopKRuntimeArgs(SizeType32 batchSize, SizeType32 topK, SizeType32* topKs, SizeType32 topKsSize,
    float topP, float* topPs, SizeType32 topPsSize, bool* skipDecode, SizeType32 const* batchSlots)
{
    auto const index = static_cast<SizeType32>(blockIdx.x * blockDim.x + threadIdx.x);
    for (auto bi = index; bi < batchSize; bi += static_cast<SizeType32>(gridDim.x * blockDim.x))
    {
        auto const batchSlot = batchSlots != nullptr ? batchSlots[bi] : bi;
        auto k = topKsSize > 1 ? topKs[batchSlot] : topK;
        auto p = topPsSize > 1 ? topPs[batchSlot] : topP;
        if (k == 0 && p == 0.0f)
        {
            // TensorRT-LLM's topp implementation does not support topp = 0.0f, but it
            // equivalent to greedy search. So, we set the topk = 1 as an alternative
            // solution.
            k = 1;
        }
        if (k > 0 && p == 0.0f)
        {
            // This case corresponds to the old topk sampling, which is equivalent to
            // the old topk_topp sampling with topp=1.0f. TopKSamplingLayer and
            // TopKTopPSamplingLayer are now merged by TopKSamplingLayer. Thus, we
            // replace the case topk>0 and topp=0.0f by topk>0 and topp=1.0f for the
            // compatibility.
            p = 1.0f;
        }
        // Clip k value. A topk sampling kernel supports up to TOP_K_MAX.
        topKs[batchSlot] = k;
        // Clip p value if it is out of range. range = [0.0, 1.0].
        topPs[batchSlot] = p;
        skipDecode[batchSlot] = k == 0;
    }
}

template <typename T>
TopKSamplingLayer<T>::TopKSamplingLayer(
    DecoderDomain const& decoderDomain, hipStream_t stream, std::shared_ptr<IAllocator> allocator)
    : BaseLayer(decoderDomain, stream, std::move(allocator))
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    allocateBuffer(mDecoderDomain.getMaxBatchSize());

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
TopKSamplingLayer<T>::~TopKSamplingLayer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    freeBuffer();

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopKSamplingLayer<T>::allocateBuffer(SizeType32 const batchSize)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    mWorkspaceSize = getTopKWorkspaceSize<T>(batchSize, 1, TOP_K_MAX, mDecoderDomain.getVocabSizePadded());

    std::array<size_t, 4> deviceBufferSizes;
    deviceBufferSizes[0] = sizeof(SizeType32) * batchSize;
    deviceBufferSizes[1] = sizeof(float) * batchSize;
    deviceBufferSizes[2] = sizeof(bool) * batchSize;
    deviceBufferSizes[3] = std::max(deviceBufferSizes[0], deviceBufferSizes[1]);

    mRuntimeTopKDevice = mAllocator->reMalloc(mRuntimeTopKDevice, deviceBufferSizes[0], false);
    mRuntimeTopPDevice = mAllocator->reMalloc(mRuntimeTopPDevice, deviceBufferSizes[1], false);
    mSkipDecodeDevice = mAllocator->reMalloc(mSkipDecodeDevice, deviceBufferSizes[2], false);
    mSetupWorkspaceDevice = mAllocator->reMalloc(mSetupWorkspaceDevice, deviceBufferSizes[3], false);

    mSkipDecodeHost = static_cast<bool*>(std::realloc(mSkipDecodeHost, sizeof(bool) * batchSize));

    mAllocatedSize = std::accumulate(deviceBufferSizes.begin(), deviceBufferSizes.end(), 0);
    TLLM_LOG_DEBUG("topKSamplingLayer allocated %lu bytes on GPU", mAllocatedSize);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopKSamplingLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    mAllocator->free((void**) (&mRuntimeTopKDevice));
    mAllocator->free((void**) (&mRuntimeTopPDevice));
    mAllocator->free((void**) (&mSkipDecodeDevice));
    mAllocator->free((void**) (&mSetupWorkspaceDevice));
    std::free(mSkipDecodeHost);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopKSamplingLayer<T>::setup(SizeType32 batchSize, SizeType32 beamWidth, SizeType32 const* batchSlots,
    std::shared_ptr<BaseSetupParams> baseSetupParams)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    auto setupParams = std::dynamic_pointer_cast<SamplingSetupParams>(baseSetupParams);

    auto const defaultTopK = DefaultDecodingParams::getTopK();
    auto runtimeTopK = setupParams->runtime_top_k.value_or(std::vector<SizeType32>(batchSize, defaultTopK));
    auto runtimeTopP = setupParams->runtime_top_p.value_or(std::vector<float>{});

    auto const runtimeTopKSize = runtimeTopK.size();
    auto const runtimeTopPSize = runtimeTopP.size();
    mNormalizeLogProbs = setupParams->normalize_log_probs.has_value() && setupParams->normalize_log_probs.value();

    for (auto& topP : runtimeTopP)
    {
        if (topP < 0.f || topP > 1.0f)
        {
            TLLM_LOG_WARNING("TopP (%f) is out of range ([0.0, 1.0f]). Clip to closest number.", topP);
            topP = std::clamp(topP, 0.f, 1.f);
        }
    }
    for (auto& topK : runtimeTopK)
    {
        if (topK < 0 || topK > TOP_K_MAX)
        {
            TLLM_LOG_WARNING(
                "TopK (%d) is larger than max supported number (%d). Clip to max supported number.", topK, TOP_K_MAX);
            topK = std::clamp(topK, 0, static_cast<SizeType32>(TOP_K_MAX));
        }
    }

    auto const topK = *std::max_element(std::begin(runtimeTopK), std::end(runtimeTopK));
    auto const topP = (runtimeTopPSize == 0) ? DefaultDecodingParams::getTopP() : runtimeTopP.front();

    if (runtimeTopKSize > 1)
    {
        TLLM_CHECK_WITH_INFO(runtimeTopK.size() == batchSize,
            fmtstr("runtimeTopK.size() (%lu) == batchSize (%d) is not satisfied!", runtimeTopK.size(), batchSize));
        cudaAutoCpy(
            reinterpret_cast<runtime::SizeType32*>(mSetupWorkspaceDevice), runtimeTopK.data(), batchSize, mStream);
        invokeScatterDecodingParams(reinterpret_cast<runtime::SizeType32*>(mSetupWorkspaceDevice), mRuntimeTopKDevice,
            batchSlots, batchSize, mStream);
    }
    if (runtimeTopPSize > 1)
    {
        TLLM_CHECK_WITH_INFO(runtimeTopP.size() == batchSize,
            fmtstr("runtimeTopP.size() (%lu) == batchSize (%d) is not satisfied!", runtimeTopP.size(), batchSize));
        cudaAutoCpy(reinterpret_cast<float*>(mSetupWorkspaceDevice), runtimeTopP.data(), batchSize, mStream);
        invokeScatterDecodingParams(
            reinterpret_cast<float*>(mSetupWorkspaceDevice), mRuntimeTopPDevice, batchSlots, batchSize, mStream);
    }

    {
        dim3 block(std::min(static_cast<uint32_t>(batchSize), 256u));
        dim3 grid(divUp(static_cast<uint32_t>(batchSize), block.x));
        // support topK up to TOP_K_MAX.
        setupTopKRuntimeArgs<TOP_K_MAX><<<grid, block, 0, mStream>>>(batchSize, topK, mRuntimeTopKDevice,
            runtimeTopKSize, topP, mRuntimeTopPDevice, runtimeTopPSize, mSkipDecodeDevice, batchSlots);
    }

    cudaAutoCpy(mSkipDecodeHost, mSkipDecodeDevice, mDecoderDomain.getMaxBatchSize(), mStream);
    std::vector<SizeType32> runtimeTopKs(mDecoderDomain.getMaxBatchSize());
    cudaAutoCpy(runtimeTopKs.data(), mRuntimeTopKDevice, mDecoderDomain.getMaxBatchSize(), mStream);
    {
        runtime::SizeType32 maxTopK = 0;
        for (SizeType32 bi = 0; bi < batchSize; ++bi)
        {
            auto bid = bi;
            if (batchSlots)
            {
                bid = batchSlots[bi];
            }
            maxTopK = std::max(maxTopK, runtimeTopKs[bid]);
        }
        mRuntimeMaxTopK = std::max(mRuntimeMaxTopK, maxTopK);
    }

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopKSamplingLayer<T>::forward(
    std::shared_ptr<BaseOutputParams> baseOutputs, std::shared_ptr<BaseInputParams> baseInputs)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    std::shared_ptr<SamplingInputParams> inputs = std::dynamic_pointer_cast<SamplingInputParams>(baseInputs);
    std::shared_ptr<SamplingOutputParams> outputs = std::dynamic_pointer_cast<SamplingOutputParams>(baseOutputs);

    auto const batchSize = inputs->logits.shape[0];

    auto logits = inputs->logits.template getPtr<T>();
    auto endIds = inputs->end_ids.template getPtr<TokenIdType const>();
    auto batchSlots = inputs->batch_slots ? inputs->batch_slots->template getPtr<SizeType32 const>() : nullptr;
    auto curandStatesDevice = inputs->curand_states;
    auto samplingWorkspaceDevice = inputs->sampling_workspace;
    auto const probsComputed = inputs->probs_computed;

    std::vector<int32_t> batchSlotsVec(batchSize);
    std::iota(batchSlotsVec.begin(), batchSlotsVec.end(), 0);
    auto batchSlotsHost
        = inputs->batch_slots ? inputs->batch_slots->template getPtr<int const>() : batchSlotsVec.data();
    auto const skip = allOfBatchSlots(batchSlotsHost, mSkipDecodeHost, batchSize, true);
    if (skip)
    {
        return;
    }

    TLLM_CHECK_WITH_INFO(curandStatesDevice, "No hiprand states provided");
    TLLM_CHECK_WITH_INFO(samplingWorkspaceDevice, "No sampling workspace provided");

    FinishedState* finishedInput = (inputs->finished)
        ? reinterpret_cast<FinishedState*>(inputs->finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;
    FinishedState* finishedOutput = (outputs->finished)
        ? reinterpret_cast<FinishedState*>(outputs->finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;

    auto cumLogProbs = (outputs->cum_log_probs) ? outputs->cum_log_probs->template getPtr<float>() : nullptr;
    auto outputLogProbs = (outputs->output_log_probs) ? outputs->output_log_probs->template getPtr<float>() : nullptr;
    auto sequenceLengths
        = (outputs->sequence_length) ? outputs->sequence_length->template getPtr<SizeType32>() : nullptr;

    TopKSamplingKernelParams<T> params;
    params.logProbs = logits;
    params.outputIdsPtrs = outputs->output_ids_ptr.template getPtr<TokenIdType*>();
    params.workspace = samplingWorkspaceDevice;
    params.maxTopP = 1.0f;
    params.topPs = mRuntimeTopPDevice;
    params.maxTopK = mRuntimeMaxTopK;
    params.topKs = mRuntimeTopKDevice;
    params.sequenceLengths = sequenceLengths;
    params.endIds = endIds;
    params.batchSlots = batchSlots;
    params.finishedInput = finishedInput;
    params.finishedOutput = finishedOutput;
    params.skipDecode = mSkipDecodeDevice;
    params.cumLogProbs = cumLogProbs;
    params.outputLogProbs = outputLogProbs;
    params.hiprandState = curandStatesDevice;
    params.batchSize = batchSize;
    params.maxBatchSize = mDecoderDomain.getMaxBatchSize();
    params.maxTokensPerStep = 1;
    params.vocabSizePadded = mDecoderDomain.getVocabSizePadded();
    params.normalizeLogProbs = mNormalizeLogProbs;
    params.logitsHasProbs = probsComputed;

    invokeBatchTopKSampling(params, mStream);
    sync_check_cuda_error();

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template class TopKSamplingLayer<float>;
template class TopKSamplingLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
