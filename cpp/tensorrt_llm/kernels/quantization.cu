#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/quantization.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

__global__ void quantizedKernel(char4* dst, float4 const* src, const int64_t sizeDiv4, float const* scalePtr)
{
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
    {
        float const scale = __ldg(scalePtr);
        char4 tmp;
        const float4 floatTmp = __ldg(src + idx);
        tmp.x = cuda_cast<int8_t>(floatTmp.x * scale);
        tmp.y = cuda_cast<int8_t>(floatTmp.y * scale);
        tmp.z = cuda_cast<int8_t>(floatTmp.z * scale);
        tmp.w = cuda_cast<int8_t>(floatTmp.w * scale);
        dst[idx] = tmp;
    }
}

__global__ void quantizedKernel(char4* dst, half2 const* src, const int64_t sizeDiv4, float const* scalePtr)
{
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
    {
        float const scale = __ldg(scalePtr);
        char4 tmp;
        int srcId = idx << 1;

        const uint2 h2 = __ldg(reinterpret_cast<uint2 const*>(src + srcId));

        const half2 half2Tmp = reinterpret_cast<half2 const&>(h2.x);
        const half2 half2Tmp2 = reinterpret_cast<half2 const&>(h2.y);

        tmp.x = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.x) * scale);
        tmp.y = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.y) * scale);
        tmp.z = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.x) * scale);
        tmp.w = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.y) * scale);
        dst[idx] = tmp;
    }
}

template <typename T>
void invokeQuantization(
    int8_t* dst, T const* src, const int64_t size, float const* scalePtr, hipStream_t stream, int maxGridSize)
{
    TLLM_CHECK_WITH_INFO(size % 4 == 0, "[ERROR][invokeQuantization] size should be a multiple of 4.\n");

    int numBlocks{static_cast<int>((size + 255) / 256)};
    dim3 grid(std::min(numBlocks, maxGridSize));
    TLLM_CHECK_WITH_INFO(grid.x <= maxGridSize, "[ERROR][invokeQuantization] grid max size is exceeded\n");
    dim3 block(64);
    if (std::is_same_v<T, float>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (float4 const*) src, size / 4, scalePtr);
    }
    else if (std::is_same_v<T, half>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (half2 const*) src, size / 4, scalePtr);
    }
}

template void invokeQuantization<float>(
    int8_t* dst, float const* src, const int64_t size, float const* scalePtr, hipStream_t stream, int maxGridSize);

template void invokeQuantization<half>(
    int8_t* dst, half const* src, const int64_t size, float const* scalePtr, hipStream_t stream, int maxGridSize);

template <typename T>
__global__ void perTokenQuantization(
    int8_t* dst, T const* src, const int64_t numRows, const int64_t numCols, float* scalePtr)
{
    T const* srcRow = src + blockIdx.x * numCols;
    int8_t* dstRow = dst + blockIdx.x * numCols;

    T localMax = 1e-6f;
    for (int i = threadIdx.x; i < numCols; i += blockDim.x)
    {
        localMax = cuda_max(localMax, cuda_abs(srcRow[i]));
    }
    float const rowMax = blockAllReduceMax(cuda_cast<float>(localMax));

    if (threadIdx.x == 0)
    {
        scalePtr[blockIdx.x] = rowMax / 127.f;
    }

    float const scaleOrigQuant = 127.f / rowMax;
    for (int i = threadIdx.x; i < numCols; i += blockDim.x)
    {
        dstRow[i] = cuda_cast<int8_t>(cuda_cast<float>(srcRow[i]) * scaleOrigQuant);
    }
}

template <typename T>
void invokePerTokenQuantization(
    int8_t* dst, T const* src, const int64_t numRows, const int64_t numCols, float* scalePtr, hipStream_t stream)
{
    // each block is responsible for a single row
    const dim3 block(512);
    const dim3 grid(numRows);

    perTokenQuantization<<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr);
}

#define INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(T)                                                                   \
    template void invokePerTokenQuantization(                                                                          \
        int8_t* dst, const T* src, const int64_t numRows, const int64_t numCols, float* scalePtr, hipStream_t stream)

INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float);
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16);
#endif

} // namespace kernels
} // namespace tensorrt_llm
