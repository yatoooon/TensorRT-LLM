#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/stopCriteriaKernels.h"

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm
{
namespace kernels
{
__global__ void stopWordsCriterion(TokenIdType const** outputIds, SizeType32 const** parentIds,
    TokenIdType const** stopWords, FinishedState* finished, SizeType32 const* sequenceLengths,
    SizeType32 const* batchSlots, SizeType32 const* stopWordsLens, SizeType32 batchSize, SizeType32 beamWidth,
    SizeType32 maxSeqLen)
{
    auto const id = static_cast<SizeType32>(blockIdx.x * blockDim.x + threadIdx.x);
    auto const batchIdx = blockIdx.y / beamWidth;
    auto const beamIdx = blockIdx.y % beamWidth;
    auto const batchSlot = batchSlots != nullptr ? batchSlots[batchIdx] : batchIdx;
    auto const batchBeamIdx = batchSlot * beamWidth + beamIdx;

    auto const* baseStopWords = stopWords[batchSlot];
    auto const stopWordsLen = stopWordsLens[batchSlot];
    auto const* baseOffsets = baseStopWords + stopWordsLen;

    if (id >= stopWordsLen || baseOffsets[id] < 0)
    {
        return;
    }

    auto const itemEnd = baseOffsets[id];
    auto const itemStart = (id > 0) ? baseOffsets[id - 1] : 0;
    auto const itemSize = itemEnd - itemStart;

    // The single-token case unconditionally bans the token
    bool shouldStop = false;

    // Need to minus 1 because the sequenceLengths is updated in this step
    auto const currentStep = sequenceLengths[batchBeamIdx] - 1;
    // Enough previously generated tokens to look for a match
    if (currentStep + 1 >= itemSize)
    {
        shouldStop = true;
        auto parentId = static_cast<SizeType32>(beamIdx);
        bool const gatherBeam = beamWidth > 1;

        for (auto tokenIdx = itemSize - 1; tokenIdx >= 0; tokenIdx--)
        {
            auto const previousToken
                = outputIds[batchSlot][parentId * maxSeqLen + currentStep - (itemSize - 1) + tokenIdx];
            if (previousToken != baseStopWords[itemStart + tokenIdx])
            {
                shouldStop = false;
                break;
            }
            if (gatherBeam)
            {
                parentId = parentIds == nullptr
                    ? SizeType32{0}
                    : parentIds[batchSlot][parentId * maxSeqLen + currentStep - (itemSize - 1) + tokenIdx];

                if (parentId < 0 || parentId >= beamWidth)
                {
                    shouldStop = false;
                    break;
                }
            }
        }
    }

    if (shouldStop)
    {
        finished[batchSlot * beamWidth + beamIdx].setFinishedStopWords();
    }
}

void invokeStopWordsCriterion(TokenIdType const** outputIds, SizeType32 const** parentIds,
    TokenIdType const** stopWords, FinishedState* finished, SizeType32 const* sequenceLengths,
    SizeType32 const* batchSlots, SizeType32 const* stopWordsLen, SizeType32 maxStopWordsLen, SizeType32 batchSize,
    SizeType32 beamWidth, SizeType32 maxSeqLen, hipStream_t stream)
{
    // Check if we have sampled a word from the stopWords list. If so, stop the sequence.
    dim3 block, grid;
    constexpr SizeType32 maxBlockSize{256};

    block.x = min(((maxStopWordsLen + 32 - 1) / 32) * 32, maxBlockSize);
    grid.x = (maxStopWordsLen + block.x - 1) / block.x;
    grid.y = batchSize * beamWidth;

    stopWordsCriterion<<<grid, block, 0, stream>>>(outputIds, parentIds, stopWords, finished, sequenceLengths,
        batchSlots, stopWordsLen, batchSize, beamWidth, maxSeqLen);
    sync_check_cuda_error();
}

__global__ void lengthCriterion(FinishedState* finished, SizeType32* finishedSum, SizeType32 const* sequenceLimitLength,
    SizeType32* sequenceLengths, SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 beamWidth)
{
    SizeType32 threadFinishedCount = 0;
    auto const batchIdx = blockIdx.x;
    auto const batchSlot = batchSlots != nullptr ? batchSlots[batchIdx] : batchIdx;

    for (auto beamIdx = static_cast<SizeType32>(threadIdx.x); beamIdx < beamWidth;
         beamIdx += static_cast<SizeType32>(blockDim.x))
    {
        auto const batchSlotBeamWidthIdx = batchSlot * beamWidth + beamIdx;

        auto finishState = finished[batchSlotBeamWidthIdx];

        if (sequenceLengths[batchSlotBeamWidthIdx] >= sequenceLimitLength[batchSlot])
        {
            finishState.setFinishedMaxLength();
            sequenceLengths[batchSlotBeamWidthIdx] = sequenceLimitLength[batchSlot];
        }
        threadFinishedCount += finishState.isFinished() ? 1 : 0;
        finished[batchSlotBeamWidthIdx] = finishState;
    }

    if (finishedSum)
    {
        SizeType32 blockFinishedCount = 0;
        if (blockDim.x <= 32)
        {
            blockFinishedCount = warpReduceSum(threadFinishedCount);
        }
        else
        {
            blockFinishedCount = blockReduceSum(threadFinishedCount);
        }
        __syncthreads();

        if (threadIdx.x == 0)
        {
            finishedSum[batchSlot] = blockFinishedCount;
        }
    }
}

void invokeLengthCriterion(FinishedState* finished, SizeType32* finishedSum, SizeType32 const* sequenceLimitLength,
    SizeType32* sequenceLengths, SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 beamWidth,
    hipStream_t stream)
{
    // Check if we have attained the sequence length limit. If so, stop the
    // sequence. In addition, check if all sequences are stopped and return the
    // result in shouldStop
    dim3 block{min(512, static_cast<uint32_t>(beamWidth))};
    dim3 grid{static_cast<uint32_t>(batchSize)};

    lengthCriterion<<<grid, block, 0, stream>>>(
        finished, finishedSum, sequenceLimitLength, sequenceLengths, batchSlots, batchSize, beamWidth);
    sync_check_cuda_error();
}

} // namespace kernels
} // namespace tensorrt_llm
