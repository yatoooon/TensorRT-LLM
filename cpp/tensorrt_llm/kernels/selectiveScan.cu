#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime_api.h>

#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>

#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>

#ifdef ENABLE_FP8
#include <hip/hip_fp8.h>
#endif

#include "selectiveScan.h"

namespace tensorrt_llm
{
namespace kernels
{

__device__ float toFloat(float f)
{
    return f;
}

__device__ float toFloat(__half h)
{
    return __half2float(h);
}
#ifdef ENABLE_BF16
__device__ float toFloat(__hip_bfloat16 val)
{
    return __bfloat162float(val);
}
#endif

__device__ void convertAndStore(float* output, float input)
{
    *output = input;
}

__device__ void convertAndStore(__half* output, float input)
{
    *output = __float2half(input);
}
#ifdef ENABLE_BF16
__device__ void convertAndStore(__hip_bfloat16* output, float input)
{
    *output = __float2bfloat16(input);
}
#endif

#pragma nv_diag_suppress static_var_with_dynamic_init

template <typename input_t, typename weight_t, int DSTATE = 16, int CHANNELS_PER_BLOCK = 128, int STAGES = 12,
    int SEQ_UNROLL = 6>
__launch_bounds__(256, 1) __global__ void selective_scan_loop_kernel(SSMParamsBase params)
{
    input_t* output = reinterpret_cast<input_t*>(params.out_ptr);
    input_t* state = reinterpret_cast<input_t*>(params.x_ptr);
    input_t* x = reinterpret_cast<input_t*>(params.u_ptr);
    input_t* dt = reinterpret_cast<input_t*>(params.delta_ptr);
    weight_t* A = reinterpret_cast<weight_t*>(params.A_ptr);
    input_t* B = reinterpret_cast<input_t*>(params.BC_ptr);
    input_t* C = reinterpret_cast<input_t*>(params.BC_ptr);
    weight_t* D = reinterpret_cast<weight_t*>(params.D_ptr);
    input_t* z = reinterpret_cast<input_t*>(params.z_ptr);
    weight_t* dt_bias = reinterpret_cast<weight_t*>(params.delta_bias_ptr);
    bool dt_softplus = params.delta_softplus;
    int num_channels = params.dim;

    __shared__ cuda::pipeline_shared_state<cuda::thread_scope::thread_scope_block, STAGES / SEQ_UNROLL> pipeline_state;
    auto block = cooperative_groups::this_thread_block();

    __shared__ __align__(16) input_t sh_B[STAGES][DSTATE];
    __shared__ __align__(16) input_t sh_C[STAGES][DSTATE];

    __shared__ __align__(128) input_t sh_dt[STAGES][CHANNELS_PER_BLOCK];
    __shared__ input_t sh_x[STAGES][CHANNELS_PER_BLOCK];
    __shared__ input_t sh_z[STAGES][CHANNELS_PER_BLOCK];

    int const channel = blockIdx.x * blockDim.x + threadIdx.x;
    int const sample = blockIdx.y; // batch id

    int const slot_idx = params.slot_mapping_ptr == nullptr ? sample : params.slot_mapping_ptr[sample];
    int const bc_cols = DSTATE * 2 + params.dt_rank;
    int const b_offset = params.dt_rank;
    int const c_offset = params.dt_rank + DSTATE;

    int num_tokens;
    int start_token_idx;
    if (params.remove_padding)
    {
        start_token_idx = sample == 0 ? 0 : params.last_token_ids_ptr[sample - 1];
        int end_token_idx = params.last_token_ids_ptr[sample];
        num_tokens = end_token_idx - start_token_idx;
    }
    else
    {
        start_token_idx = sample * params.max_seqlen;
        num_tokens = params.last_token_ids_ptr[sample];
    }
    int const seq_loops = (num_tokens + SEQ_UNROLL - 1) / SEQ_UNROLL;

    int const input_matrix_row_id = start_token_idx;

    if (threadIdx.y == 1)
    {
        cuda::pipeline pipeline = cuda::make_pipeline(block, &pipeline_state, cuda::pipeline_role::producer);

        int stage = 0;
        for (int si = 0; si < seq_loops; si++)
        {

            pipeline.producer_acquire();

#pragma unroll
            for (int token_id = si * SEQ_UNROLL; token_id < num_tokens && token_id < (si + 1) * SEQ_UNROLL; token_id++)
            {

                input_t* my_B = &B[(input_matrix_row_id + token_id) * bc_cols + b_offset];
                input_t* my_C = &C[(input_matrix_row_id + token_id) * bc_cols + c_offset];

                int block_channel_per_token = blockIdx.x * blockDim.x;
                int block_channel
                    = input_matrix_row_id * num_channels + token_id * num_channels + block_channel_per_token;

                if (threadIdx.x < DSTATE)
                    cuda::memcpy_async(&sh_B[stage][threadIdx.x], &my_B[threadIdx.x], sizeof(input_t), pipeline);
                else if (threadIdx.x >= 32 && threadIdx.x < 32 + DSTATE)
                    cuda::memcpy_async(
                        &sh_C[stage][threadIdx.x - 32], &my_C[threadIdx.x - 32], sizeof(input_t), pipeline);
                if (sizeof(input_t) == 4)
                {
                    cuda::memcpy_async(&sh_dt[stage][threadIdx.x],
                        &dt[input_matrix_row_id * num_channels + token_id * num_channels + channel], sizeof(input_t),
                        pipeline);
                    cuda::memcpy_async(&sh_x[stage][threadIdx.x],
                        &x[input_matrix_row_id * num_channels + token_id * num_channels + channel], sizeof(input_t),
                        pipeline);
                    if (z)
                        cuda::memcpy_async(&sh_z[stage][threadIdx.x],
                            &z[input_matrix_row_id * num_channels + token_id * num_channels + channel], sizeof(input_t),
                            pipeline);
                }
                else
                {
                    // sh_dt[stage][threadIdx.x] = dt[block_channel + threadIdx.x];
                    if (threadIdx.x < 32)
                    {
                        int tid = threadIdx.x;
                        float2* block_dt = (float2*) &dt[block_channel];
                        cuda::memcpy_async((float2*) &sh_dt[stage][tid * 4], &block_dt[tid], sizeof(float2), pipeline);
                    }
                    // sh_x[stage][threadIdx.x] = x[block_channel + threadIdx.x];
                    else if (threadIdx.x < 64)
                    {
                        int tid = threadIdx.x - 32;
                        float2* block_x = (float2*) &x[block_channel];
                        cuda::memcpy_async((float2*) &sh_x[stage][tid * 4], &block_x[tid], sizeof(float2), pipeline);
                    }
                    // sh_z[stage][threadIdx.x] = z[block_channel + threadIdx.x];
                    else if (threadIdx.x < 96)
                    {
                        int tid = threadIdx.x - 64;
                        if (z)
                        {
                            float2* block_z = (float2*) &z[block_channel];
                            cuda::memcpy_async(
                                (float2*) &sh_z[stage][tid * 4], &block_z[tid], sizeof(float2), pipeline);
                        }
                    }
                    else
                    {
                    }
                }

                stage++;
                if (stage >= STAGES)
                    stage = 0;
            }
            pipeline.producer_commit();
        }
    }
    else
    {

        // Compute warps
        // Load state and A matrix into registers
        float state_reg[DSTATE];
        float A_reg[DSTATE];
        for (int i = 0; i < DSTATE; i++)
        {
            state_reg[i] = 0.f;
            A_reg[i] = toFloat(A[i * num_channels + channel]);
        }
        float dt_bias_reg = dt_bias[channel];
        float D_reg = D ? D[channel] : 0.f;

        cuda::pipeline pipeline = cuda::make_pipeline(block, &pipeline_state, cuda::pipeline_role::consumer);
        int stage = 0;
        for (int si = 0; si < seq_loops; si++)
        {

            pipeline.consumer_wait();

#pragma unroll
            for (int token_id = si * SEQ_UNROLL; token_id < num_tokens && token_id < (si + 1) * SEQ_UNROLL; token_id++)
            {

                float dt_b = toFloat(sh_dt[stage][threadIdx.x]) + dt_bias_reg;
                float dt_b_sp;
                if (dt_softplus)
                {
                    dt_b_sp = dt_b <= 20.f ? __logf(1.f + __expf(dt_b)) : dt_b; // softplus
                }
                float my_x = toFloat(sh_x[stage][threadIdx.x]);
                float Dx = my_x * D_reg;
                float dtx = dt_b_sp * my_x;
                float my_z = z ? toFloat(sh_z[stage][threadIdx.x]) : 0.f;

                float out = Dx;

                if (sizeof(input_t) == 4)
                {
                    float4* B4 = (float4*) &sh_B[stage][0];
                    float4* C4 = (float4*) &sh_C[stage][0];
#pragma unroll
                    for (int i = 0; i < DSTATE / 4; i++)
                    {

                        float4 Bi4 = B4[i];
                        float4 Ci4 = C4[i];

                        float* Bi = (float*) &Bi4;
                        float* Ci = (float*) &Ci4;

#pragma unroll
                        for (int j = 0; j < 4; j++)
                        {
                            float dtA = A_reg[i * 4 + j] * dt_b_sp;
                            float dA = __expf(dtA);
                            float sdA = state_reg[i * 4 + j] * dA;
                            float dBx = Bi[j] * dtx;
                            float newState = sdA + dBx;
                            state_reg[i * 4 + j] = newState;
                            out += newState * Ci[j];
                        }
                    }
                }
                else
                {
                    float4* B8 = (float4*) &sh_B[stage][0];
                    float4* C8 = (float4*) &sh_C[stage][0];
#pragma unroll
                    for (int i = 0; i < DSTATE / 8; i++)
                    {
                        input_t* Bi = (input_t*) (&B8[i]);
                        input_t* Ci = (input_t*) (&C8[i]);
#pragma unroll
                        for (int j = 0; j < 8; j++)
                        {
                            float dtA = A_reg[i * 8 + j] * dt_b_sp;
                            float dA = __expf(dtA);
                            float sdA = state_reg[i * 8 + j] * dA;
                            float dBx = toFloat(Bi[j]) * dtx;
                            float newState = sdA + dBx;
                            state_reg[i * 8 + j] = newState;
                            out += newState * toFloat(Ci[j]);
                        }
                    }
                }

                if (z)
                {
                    float enz = __expf(0.f - my_z);
                    enz += 1.0;
                    float sig_z = __fdividef(1.f, enz);
                    float silu_z = my_z * sig_z;
                    out *= silu_z;
                }
                input_t* my_output = &output[input_matrix_row_id * num_channels + token_id * num_channels];
                convertAndStore(&my_output[channel], out);

                stage++;
                if (stage >= STAGES)
                    stage = 0;
            }
            pipeline.consumer_release();
        }
        // Write the new state back out to the cache
        for (int i = 0; i < DSTATE; i++)
        {
            input_t* my_state = &state[slot_idx * num_channels * DSTATE];
            int offset = i * num_channels + channel;
            convertAndStore(&my_state[offset], state_reg[i]);
        }
    }
}

template <typename input_t, typename weight_t>
void invokeSelectiveScan(SSMParamsBase& params, hipStream_t stream)
{
    int samples = params.batch;
    int channels = params.dim;

    TLLM_CHECK(params.is_variable_B);
    TLLM_CHECK(params.is_variable_C);
    TLLM_CHECK(params.dstate == 16);

    int const threads = 128;
    int const blocks = (channels + threads - 1) / threads;
    dim3 block(threads, 2);
    dim3 grid(blocks, samples);
    TLLM_CHECK((channels % block.x) == 0);
    selective_scan_loop_kernel<input_t, weight_t><<<grid, block, 0, stream>>>(params);
}

#define INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(input_t, weight_t)                                                        \
    template void invokeSelectiveScan<input_t, weight_t>(SSMParamsBase & params, hipStream_t stream);

INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(float, float);
INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(half, float);
#ifdef ENABLE_BF16
INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(__hip_bfloat16, float);
#endif
#undef INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename input_t, typename weight_t, int DSTATE = 16, int CHANNELS_PER_BLOCK = 128>
__launch_bounds__(128, 2) __global__ void selective_scan_update_kernel(SSMParamsBase params)
{

    input_t* output = reinterpret_cast<input_t*>(params.out_ptr);
    input_t* state = reinterpret_cast<input_t*>(params.x_ptr);
    input_t* x = reinterpret_cast<input_t*>(params.u_ptr);
    input_t* dt = reinterpret_cast<input_t*>(params.delta_ptr);
    weight_t* A = reinterpret_cast<weight_t*>(params.A_ptr);
    input_t* B = reinterpret_cast<input_t*>(params.BC_ptr);
    input_t* C = reinterpret_cast<input_t*>(params.BC_ptr);
    weight_t* D = reinterpret_cast<weight_t*>(params.D_ptr);
    input_t* z = reinterpret_cast<input_t*>(params.z_ptr);
    weight_t* dt_bias = reinterpret_cast<weight_t*>(params.delta_bias_ptr);
    bool dt_softplus = params.delta_softplus;
    int num_channels = params.dim;

    int const channel = blockIdx.x * blockDim.x + threadIdx.x;
    if (channel >= num_channels)
        return;
    int const sample = blockIdx.y;
    int const slot_idx = params.slot_mapping_ptr == nullptr ? sample : params.slot_mapping_ptr[sample];
    int const bc_cols = DSTATE * 2 + params.dt_rank;
    int const b_offset = params.dt_rank;
    int const c_offset = params.dt_rank + DSTATE;

    input_t* my_state = &state[slot_idx * num_channels * DSTATE];
    input_t* my_output = &output[sample * num_channels];

    float rA[DSTATE];
    float rB[DSTATE];
    float rC[DSTATE];

    float rState[DSTATE];

#pragma unroll
    for (int i = 0; i < DSTATE; i++)
    {
        rA[i] = toFloat(A[i * num_channels + channel]);
        rB[i] = toFloat(B[sample * bc_cols + b_offset + i]);
        rC[i] = toFloat(C[sample * bc_cols + c_offset + i]);
        rState[i] = toFloat(my_state[i * num_channels + channel]);
    }

    float my_x, my_dt, my_z, my_dt_bias, my_D;
    my_x = toFloat(x[sample * num_channels + channel]);
    my_dt = toFloat(dt[sample * num_channels + channel]);
    my_z = z ? toFloat(z[sample * num_channels + channel]) : 0.f;
    my_dt_bias = dt_bias ? toFloat(dt_bias[channel]) : 0.f;
    my_D = D ? toFloat(D[channel]) : 0.f;

    float dt_b = my_dt + my_dt_bias;
    float dt_b_sp;
    if (dt_softplus)
    {
        // dt_b_sp = dt_b <= 20.f ? logf(1.f + expf(dt_b)) : dt_b; // softplus
        dt_b_sp = dt_b <= 20.f ? __logf(1.f + __expf(dt_b)) : dt_b; // softplus
    }

    float out = D ? my_D * my_x : 0.f;

#pragma unroll
    for (int i = 0; i < DSTATE; i++)
    {
        // float dA = expf(rA[i] * dt_b_sp);
        float dA = __expf(rA[i] * dt_b_sp);
        float dB = rB[i] * dt_b_sp;
        float sdA = rState[i] * dA;
        float dBx = dB * my_x;
        float newState = sdA + dBx;
        convertAndStore(&my_state[i * num_channels + channel], newState); // Write the new state back out to the cache
        out += newState * rC[i];
    }

    if (z)
    {
        // float sig_z = 1.0 / (1.0 + exp(0.f - my_z));
        float sig_z = __fdividef(1.f, (1.f + __expf(0.f - my_z)));
        float silu_z = my_z * sig_z;
        out *= silu_z;
    }

    convertAndStore(&my_output[channel], out);
}

template <typename input_t, typename weight_t>
void invokeSelectiveScanUpdate(SSMParamsBase& params, hipStream_t stream)
{
    int samples = params.batch;
    int channels = params.dim;

    int const threads = 128;
    int const blocks = (channels + threads - 1) / threads;
    dim3 block(threads, 1);
    dim3 grid(blocks, samples);

    TLLM_CHECK(params.is_variable_B);
    TLLM_CHECK(params.is_variable_C);
    TLLM_CHECK(params.dstate == 16);
    selective_scan_update_kernel<input_t, weight_t><<<grid, block, 0, stream>>>(params);
}

#define INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(input_t, weight_t)                                                 \
    template void invokeSelectiveScanUpdate<input_t, weight_t>(SSMParamsBase & params, hipStream_t stream)

INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(float, float);
INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(half, float);
#ifdef ENABLE_BF16
INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(__hip_bfloat16, float);
#endif
#undef INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE

} // namespace kernels
} // namespace tensorrt_llm
