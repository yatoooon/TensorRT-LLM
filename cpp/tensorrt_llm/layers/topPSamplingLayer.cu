#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include "tensorrt_llm/kernels/samplingTopKKernels.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"
#include "tensorrt_llm/layers/defaultDecodingParams.h"
#include "tensorrt_llm/layers/layerUtils.h"
#include "tensorrt_llm/layers/topPSamplingLayer.h"

#include <algorithm>
#include <float.h>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm
{
namespace layers
{

static __global__ void setTopPRuntimeArgs(SizeType32 batchSize, SizeType32 topK, SizeType32* topKs,
    SizeType32 topKsSize, float topP, float* topPs, SizeType32 topPsSize, bool* skipDecode,
    SizeType32 const* batchSlots, float* initialTopPBuf)
{
    /**
     * @brief Setup the runtime arguments for topp, broadcasting top_p to top_ps
              and top_k to top_ks.
     */

    auto index = static_cast<SizeType32>(blockIdx.x * blockDim.x + threadIdx.x);
    for (SizeType32 bi = index; bi < batchSize; bi += static_cast<SizeType32>(gridDim.x * blockDim.x))
    {
        auto const batchSlot = batchSlots != nullptr ? batchSlots[bi] : bi;
        auto k = topKsSize > 1 ? topKs[batchSlot] : topK;
        auto const p = topPsSize > 1 ? topPs[batchSlot] : topP;
        if (k == 0 && p == 0.0f)
        {
            // TensorRT-LLM's topp implementation does not support topp = 0.0f, but it
            // equivalent to greedy search. So, we set the topk = 1 as an alternative
            // solution.
            k = 1;
        }
        topKs[batchSlot] = k;
        topPs[batchSlot] = p;
        skipDecode[batchSlot] = k > 0;

        initialTopPBuf[batchSlot] = topPs[batchSlot];
    }
}

template <typename T>
TopPSamplingLayer<T>::TopPSamplingLayer(DecoderDomain const& decoderDomain, hipStream_t stream,
    std::shared_ptr<IAllocator> allocator, bool isDeterministic, bool isAirTopP)
    : BaseLayer(decoderDomain, stream, std::move(allocator))
    , mIsDeterministic(isDeterministic)
    , mIsAirTopP(isAirTopP)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    int deviceId;
    tc::check_cuda_error(hipGetDevice(&deviceId)); // Get the correct device id
    tc::check_cuda_error(hipGetDeviceProperties(&mDeviceProp, deviceId));

    allocateBuffer(mDecoderDomain.getMaxBatchSize());

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
TopPSamplingLayer<T>::~TopPSamplingLayer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    freeBuffer();

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopPSamplingLayer<T>::allocateBuffer(SizeType32 batchSize)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    if (mIsAirTopP == false)
    {
        mWorkspaceSize = getTopPWorkspaceSize<T>(batchSize, mDecoderDomain.getVocabSizePadded());
    }
    else
    {
        mWorkspaceSize = getAirTopPWorkspaceSize<T>(batchSize, mDecoderDomain.getVocabSizePadded(), mIsDeterministic);
    }

    std::array<size_t, 8> deviceBufferSizes = {sizeof(SizeType32) * batchSize, sizeof(float) * batchSize,
        sizeof(float) * batchSize, sizeof(float) * batchSize, sizeof(float) * batchSize,
        sizeof(TokenIdType) * batchSize, sizeof(bool) * batchSize, 0};
    deviceBufferSizes[7] = *std::max_element(&deviceBufferSizes[0], &deviceBufferSizes[6]);

    mRuntimeTopKDevice = mAllocator->reMalloc(mRuntimeTopKDevice, deviceBufferSizes[0], false);
    mRuntimeTopPDevice = mAllocator->reMalloc(mRuntimeTopPDevice, deviceBufferSizes[1], false);
    mInitialTopPDevice = mAllocator->reMalloc(mInitialTopPDevice, deviceBufferSizes[2], false);
    mTopPDecayDevice = mAllocator->reMalloc(mTopPDecayDevice, deviceBufferSizes[3], false);
    mTopPMinDevice = mAllocator->reMalloc(mTopPMinDevice, deviceBufferSizes[4], false);
    mTopPResetIdsDevice = mAllocator->reMalloc(mTopPResetIdsDevice, deviceBufferSizes[5], false);
    mSkipDecodeDevice = mAllocator->reMalloc(mSkipDecodeDevice, deviceBufferSizes[6], false);
    mSetupWorkspaceDevice = mAllocator->reMalloc(mSetupWorkspaceDevice, deviceBufferSizes[7], false);

    mSkipDecodeHost = static_cast<bool*>(std::realloc(mSkipDecodeHost, sizeof(bool) * batchSize));
    std::fill(mSkipDecodeHost, mSkipDecodeHost + batchSize, true);

    mAllocatedSize = std::accumulate(deviceBufferSizes.begin(), deviceBufferSizes.end(), 0);
    TLLM_LOG_DEBUG("topPSamplingLayer allocated %lu bytes on GPU", mAllocatedSize);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopPSamplingLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    mAllocator->free((void**) (&mRuntimeTopKDevice));
    mAllocator->free((void**) (&mRuntimeTopPDevice));
    mAllocator->free((void**) (&mInitialTopPDevice));
    mAllocator->free((void**) (&mTopPDecayDevice));
    mAllocator->free((void**) (&mTopPMinDevice));
    mAllocator->free((void**) (&mTopPResetIdsDevice));
    mAllocator->free((void**) (&mSkipDecodeDevice));
    mAllocator->free((void**) (&mSetupWorkspaceDevice));
    std::free(mSkipDecodeHost);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopPSamplingLayer<T>::setup(SizeType32 const batchSize, SizeType32 const beamWidth, SizeType32 const* batchSlots,
    std::shared_ptr<BaseSetupParams> baseSetupParams)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    auto setupParams = std::dynamic_pointer_cast<SamplingSetupParams>(baseSetupParams);

    auto const defaultTopK = DefaultDecodingParams::getTopK();
    auto runtimeTopK = setupParams->runtime_top_k.value_or(std::vector<SizeType32>(batchSize, defaultTopK));
    auto runtimeTopP = setupParams->runtime_top_p.value_or(std::vector<float>{});

    auto const runtimeTopKSize = runtimeTopK.size();
    auto const runtimeTopPSize = runtimeTopP.size();

    auto const defaultTopPDecay = DefaultDecodingParams::getTopPDecay();
    auto decayVec = setupParams->top_p_decay.value_or(std::vector<float>(batchSize, defaultTopPDecay));

    auto const defaultTopPMin = DefaultDecodingParams::getTopPMin(); // prevent TopP becoming 0.0
    auto topPMinVec = setupParams->top_p_min.value_or(std::vector<float>(batchSize, defaultTopPMin));

    auto const defaultTopPResetId = DefaultDecodingParams::getTopPResetId();
    auto topPResetIdsVec
        = setupParams->top_p_reset_ids.value_or(std::vector<TokenIdType>(batchSize, defaultTopPResetId));

    if (runtimeTopPSize == 0)
    {
        for (SizeType32 bi = 0; bi < batchSize; ++bi)
        {
            auto bid = bi;
            if (batchSlots)
            {
                bid = batchSlots[bi];
            }
            mSkipDecodeHost[bid] = true;
        }
        cudaAutoCpy(mSkipDecodeDevice, mSkipDecodeHost, mDecoderDomain.getMaxBatchSize(), mStream);
        return;
    }

    for (auto& topP : runtimeTopP)
    {
        if (topP < 0.f || topP > 1.0f)
        {
            TLLM_LOG_WARNING("TopP (%f) is out of range ([0.0, 1.0f]). Clip to closest number.", topP);
            topP = std::clamp(topP, 0.f, 1.f);
        }
    }

    for (auto& decay : decayVec)
    {
        if (decay <= 0.f || decay > 1.0f)
        {
            TLLM_LOG_WARNING(
                "Decay (%f) is out of range ((0.0, 1.0f]). Change to default (%f).", decay, defaultTopPDecay);
            decay = defaultTopPDecay;
        }
    }

    for (auto& topPMin : topPMinVec)
    {
        if (topPMin <= 0.f || topPMin > 1.0f)
        {
            TLLM_LOG_WARNING(
                "TopP min (%f) is out of range ([0.0, 1.0f]). Change to default (%f).", topPMin, defaultTopPMin);
            topPMin = defaultTopPMin;
        }
    }

    auto const topK = runtimeTopK.at(0);
    auto const topP = runtimeTopP.at(0);

    if (runtimeTopKSize > 1)
    {
        TLLM_CHECK_WITH_INFO(static_cast<SizeType32>(runtimeTopK.size()) == batchSize,
            fmtstr("runtimeTopK.size() (%lu) == batchSize (%d) is not satisfied!", runtimeTopK.size(), batchSize));
        cudaAutoCpy(reinterpret_cast<SizeType32*>(mSetupWorkspaceDevice), runtimeTopK.data(), batchSize, mStream);
        invokeScatterDecodingParams(
            reinterpret_cast<SizeType32*>(mSetupWorkspaceDevice), mRuntimeTopKDevice, batchSlots, batchSize, mStream);
    }
    if (runtimeTopPSize > 1)
    {
        TLLM_CHECK_WITH_INFO(static_cast<SizeType32>(runtimeTopP.size()) == batchSize,
            fmtstr("runtime_top_p.size() (%lu) == batchSize (%d) is not satisfied!", runtimeTopP.size(), batchSize));
        cudaAutoCpy(reinterpret_cast<float*>(mSetupWorkspaceDevice), runtimeTopP.data(), batchSize, mStream);
        invokeScatterDecodingParams(
            reinterpret_cast<float*>(mSetupWorkspaceDevice), mRuntimeTopPDevice, batchSlots, batchSize, mStream);
    }

    auto fillBuffers
        = [this, &batchSize, &batchSlots](std::string name, auto const& vector, auto deviceTmpBuffer, auto deviceBuffer)
    {
        TLLM_CHECK_WITH_INFO(static_cast<SizeType32>(vector.size()) == batchSize,
            fmtstr("%s.size() (%lu) == batchSize (%d) is not satisfied!", name.c_str(), vector.size(), batchSize));
        cudaAutoCpy(deviceTmpBuffer, vector.data(), batchSize, mStream);
        invokeScatterDecodingParams(deviceTmpBuffer, deviceBuffer, batchSlots, batchSize, mStream);
    };

    fillBuffers("top_p_decay", decayVec, reinterpret_cast<float*>(mSetupWorkspaceDevice), mTopPDecayDevice);

    fillBuffers("top_p_min", topPMinVec, reinterpret_cast<float*>(mSetupWorkspaceDevice), mTopPMinDevice);

    fillBuffers(
        "top_p_reset_ids", topPResetIdsVec, reinterpret_cast<TokenIdType*>(mSetupWorkspaceDevice), mTopPResetIdsDevice);

    {
        dim3 block(std::min(static_cast<uint32_t>(batchSize), 256u));
        dim3 grid(divUp(static_cast<uint32_t>(batchSize), block.x));
        setTopPRuntimeArgs<<<grid, block, 0, mStream>>>(batchSize, topK, mRuntimeTopKDevice, runtimeTopKSize, topP,
            mRuntimeTopPDevice, runtimeTopPSize, mSkipDecodeDevice, batchSlots, mInitialTopPDevice);
        sync_check_cuda_error();
    }

    cudaAutoCpy(mSkipDecodeHost, mSkipDecodeDevice, mDecoderDomain.getMaxBatchSize(), mStream);
    std::vector<float> runtimeTopPs(mDecoderDomain.getMaxBatchSize());
    cudaAutoCpy(runtimeTopPs.data(), mRuntimeTopPDevice, mDecoderDomain.getMaxBatchSize(), mStream);
    {
        auto maxTopP = 0.f;
        for (SizeType32 bi = 0; bi < batchSize; ++bi)
        {
            auto const bid = batchSlots ? batchSlots[bi] : bi;
            maxTopP = std::max(maxTopP, runtimeTopPs[bid]);
        }
        mRuntimeMaxTopP = std::max(mRuntimeMaxTopP, maxTopP);
    }

    if (mIsAirTopP == true)
    {
        auto smCnt = mDeviceProp.multiProcessorCount;
        if (smCnt <= 0)
        {
            int deviceId;
            check_cuda_error(hipGetDevice(&deviceId)); // Get the correct device id
            hipDeviceProp_t prop;
            check_cuda_error(hipGetDeviceProperties(&prop, deviceId));
            smCnt = prop.multiProcessorCount;
        }
        mAirTopPBlockNum
            = calcAirTopPBlockNum<T>(batchSize, (int) mDecoderDomain.getVocabSizePadded(), smCnt, mIsDeterministic);
    }

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void TopPSamplingLayer<T>::forward(
    std::shared_ptr<BaseOutputParams> baseOutputs, std::shared_ptr<BaseInputParams> baseInputs)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    std::shared_ptr<SamplingInputParams> inputs = std::dynamic_pointer_cast<SamplingInputParams>(baseInputs);
    std::shared_ptr<SamplingOutputParams> outputs = std::dynamic_pointer_cast<SamplingOutputParams>(baseOutputs);

    auto const batchSize = inputs->logits.shape[0];

    std::vector<int32_t> batchSlotsVec(batchSize);
    std::iota(batchSlotsVec.begin(), batchSlotsVec.end(), 0);
    auto batchSlotsHost
        = inputs->batch_slots ? inputs->batch_slots->template getPtr<int const>() : batchSlotsVec.data();
    auto const skip = allOfBatchSlots(batchSlotsHost, mSkipDecodeHost, batchSize, true);
    if (skip)
    {
        return;
    }

    // Probabilities must be already computed instead of logits
    auto probs = inputs->logits.template getPtr<T>();
    auto endIds = inputs->end_ids.template getPtr<TokenIdType const>();
    auto batchSlots = inputs->batch_slots ? inputs->batch_slots->template getPtr<SizeType32 const>() : nullptr;
    auto curandStatesDevice = inputs->curand_states;
    auto samplingWorkspaceDevice = inputs->sampling_workspace;

    TLLM_CHECK_WITH_INFO(curandStatesDevice, "No hiprand states provided");
    TLLM_CHECK_WITH_INFO(samplingWorkspaceDevice, "No sampling workspace provided");

    FinishedState* finishedInput = (inputs->finished)
        ? reinterpret_cast<FinishedState*>(inputs->finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;
    FinishedState* finishedOutput = (outputs->finished)
        ? reinterpret_cast<FinishedState*>(outputs->finished->template getPtr<FinishedState::UnderlyingType>())
        : nullptr;

    auto cumLogProbs = (outputs->cum_log_probs) ? outputs->cum_log_probs->template getPtr<float>() : nullptr;
    auto outputLogProbs = (outputs->output_log_probs) ? outputs->output_log_probs->template getPtr<float>() : nullptr;
    auto sequenceLength
        = (outputs->sequence_length) ? outputs->sequence_length->template getPtr<SizeType32>() : nullptr;

    TopPSamplingKernelParams<T> params;
    params.probs = probs;
    params.outputIds = outputs->output_ids_ptr.template getPtr<TokenIdType*>();
    params.workspace = samplingWorkspaceDevice;
    params.topPs = mRuntimeTopPDevice;
    params.sequenceLength = sequenceLength;
    params.endIds = endIds;
    params.batchSlots = batchSlots;
    params.finishedInput = finishedInput;
    params.finishedOutput = finishedOutput;
    params.skipDecode = mSkipDecodeDevice;
    params.cumLogProbs = cumLogProbs;
    params.outputLogProbs = outputLogProbs;
    params.hiprandState = curandStatesDevice;
    params.batchSize = batchSize;
    params.maxBatchSize = mDecoderDomain.getMaxBatchSize();
    params.vocabSizePadded = mDecoderDomain.getVocabSizePadded();

    if (mIsAirTopP == false)
    {
        invokeBatchTopPSampling<T>(params, mStream);
        sync_check_cuda_error();
    }
    else
    {
        params.blockNum = mAirTopPBlockNum;
        params.isDeterministic = mIsDeterministic;
        invokeBatchAirTopPSampling<T>(params, mStream);
        sync_check_cuda_error();
    }

    sync_check_cuda_error();
    invokeComputeToppDecay(mRuntimeTopPDevice, mInitialTopPDevice,
        outputs->output_ids_ptr.template getPtr<TokenIdType const*>(), mTopPDecayDevice, mTopPMinDevice,
        mTopPResetIdsDevice, sequenceLength, batchSlots, batchSize, mStream);
    sync_check_cuda_error();
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template class TopPSamplingLayer<float>;
template class TopPSamplingLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
