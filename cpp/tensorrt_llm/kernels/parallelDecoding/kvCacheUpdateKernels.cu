#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "kvCacheUpdateKernels.h"

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/kernels/kvCacheUtils.h"

#include <array>
#include <vector>

namespace tensorrt_llm::kernels::parallel_decoding
{

static constexpr int kUpdateKVCacheKernelShmSize = 16384;

template <typename KVCacheBuffer, int MaxLayerCount, typename MoveEltType>
__global__ void updateKVCacheDraftTokenLocationBatchedKernel(std::array<KVCacheBuffer, MaxLayerCount> kvCacheBuffers,
    int const* seqAcceptedDraftTokenOffsets, IndexType const* packedAcceptedDraftTokensIndices,
    int32_t const* pastKeyValueLengths, int rewindDraftTokenCommonCount, int const* rewindDraftTokenSeparateAdjustments,
    int const* seqSlotRemapping, int eltCountPerHead)
{
    int seqIdx = blockIdx.x;
    int headIdx = blockIdx.y;
    int layerIdx = blockIdx.z;
    int warpIdx = threadIdx.x / 32;
    int warpCount = blockDim.x / 32;
    int laneIdx = threadIdx.x & 0x1f;
    int seqDraftTokenStart = seqAcceptedDraftTokenOffsets[seqIdx];
    int seqDraftTokenEnd = seqAcceptedDraftTokenOffsets[seqIdx + 1];
    auto const seqSlot = seqSlotRemapping == nullptr ? seqIdx : seqSlotRemapping[seqIdx];
    int seqDraftCount = seqDraftTokenEnd - seqDraftTokenStart;
    int maxEltCountPerMove = kUpdateKVCacheKernelShmSize / sizeof(MoveEltType) / seqDraftCount;
    int eltCountPerMove = min(maxEltCountPerMove, eltCountPerHead);
    if (seqDraftCount == 0 || eltCountPerMove == 0)
    {
        return;
    }
    KVCacheBuffer& kvCacheBuffer = kvCacheBuffers[layerIdx];
    int tokenStartIdx = pastKeyValueLengths[seqSlot] - rewindDraftTokenCommonCount;
    if (rewindDraftTokenSeparateAdjustments != nullptr)
    {
        tokenStartIdx -= rewindDraftTokenSeparateAdjustments[seqSlot];
    }
    __shared__ char loadSmemBuffer[kUpdateKVCacheKernelShmSize];
    auto* eltLoadSmemBuffer = reinterpret_cast<MoveEltType*>(&loadSmemBuffer[0]);
    for (int startChannelOffset = 0; startChannelOffset < eltCountPerHead; startChannelOffset += eltCountPerMove)
    {
        int eltCountCurrentMove = min(eltCountPerMove, eltCountPerHead - startChannelOffset);
        // load K
        for (int tokenIdx = warpIdx; tokenIdx < seqDraftCount; tokenIdx += warpCount)
        {
            int tokenPos = packedAcceptedDraftTokensIndices[seqDraftTokenStart + tokenIdx];
            auto* tokenSmemBuffer = eltLoadSmemBuffer + tokenIdx * eltCountCurrentMove;
            int tokenKVPosition = tokenStartIdx + tokenPos;
            auto* kPtr = reinterpret_cast<MoveEltType*>(kvCacheBuffer.getKBlockPtr(seqSlot, tokenKVPosition));
            for (int loadChannelIdx = laneIdx; loadChannelIdx < eltCountCurrentMove; loadChannelIdx += 32)
            {
                int channelIdx = loadChannelIdx + startChannelOffset;
                int kvLocationIdx = kvCacheBuffer.getKVLocalIdx(tokenKVPosition, headIdx, eltCountPerHead, channelIdx);
                tokenSmemBuffer[loadChannelIdx] = kPtr[kvLocationIdx];
            }
        }
        __syncthreads();
        // store K
        for (int tokenIdx = warpIdx; tokenIdx < seqDraftCount; tokenIdx += warpCount)
        {
            int tokenPos = tokenIdx;
            auto* tokenSmemBuffer = eltLoadSmemBuffer + tokenIdx * eltCountCurrentMove;
            int tokenKVPosition = tokenStartIdx + tokenPos;
            auto* kPtr = reinterpret_cast<MoveEltType*>(kvCacheBuffer.getKBlockPtr(seqSlot, tokenKVPosition));
            for (int loadChannelIdx = laneIdx; loadChannelIdx < eltCountCurrentMove; loadChannelIdx += 32)
            {
                int channelIdx = loadChannelIdx + startChannelOffset;
                int kvLocationIdx = kvCacheBuffer.getKVLocalIdx(tokenKVPosition, headIdx, eltCountPerHead, channelIdx);
                kPtr[kvLocationIdx] = tokenSmemBuffer[loadChannelIdx];
            }
        }
        __syncthreads();
        // load V
        for (int tokenIdx = warpIdx; tokenIdx < seqDraftCount; tokenIdx += warpCount)
        {
            int tokenPos = packedAcceptedDraftTokensIndices[seqDraftTokenStart + tokenIdx];
            auto* tokenSmemBuffer = eltLoadSmemBuffer + tokenIdx * eltCountCurrentMove;
            int tokenKVPosition = tokenStartIdx + tokenPos;
            auto* vPtr = reinterpret_cast<MoveEltType*>(kvCacheBuffer.getVBlockPtr(seqSlot, tokenKVPosition));
            for (int loadChannelIdx = laneIdx; loadChannelIdx < eltCountCurrentMove; loadChannelIdx += 32)
            {
                int channelIdx = loadChannelIdx + startChannelOffset;
                int kvLocationIdx = kvCacheBuffer.getKVLocalIdx(tokenKVPosition, headIdx, eltCountPerHead, channelIdx);
                tokenSmemBuffer[loadChannelIdx] = vPtr[kvLocationIdx];
            }
        }
        __syncthreads();
        // store V
        for (int tokenIdx = warpIdx; tokenIdx < seqDraftCount; tokenIdx += warpCount)
        {
            int tokenPos = tokenIdx;
            auto* tokenSmemBuffer = eltLoadSmemBuffer + tokenPos * eltCountCurrentMove;
            int tokenKVPosition = tokenStartIdx + tokenPos;
            auto* vPtr = reinterpret_cast<MoveEltType*>(kvCacheBuffer.getVBlockPtr(seqSlot, tokenKVPosition));
            for (int loadChannelIdx = laneIdx; loadChannelIdx < eltCountCurrentMove; loadChannelIdx += 32)
            {
                int channelIdx = loadChannelIdx + startChannelOffset;
                int kvLocationIdx = kvCacheBuffer.getKVLocalIdx(tokenKVPosition, headIdx, eltCountPerHead, channelIdx);
                vPtr[kvLocationIdx] = tokenSmemBuffer[loadChannelIdx];
            }
        }
        __syncthreads();
    }
}

template <typename KVCacheBuffer, int MaxLayerCount>
void updateKVCacheDraftTokenLocationBatched(KVCacheBuffer const* kvCacheBuffers,
    int const* seqAcceptedDraftTokenOffsets, IndexType const* packedAcceptedDraftTokensIndices,
    int32_t const* pastKeyValueLengths, int layerCount, int seqCount, int numKVHeads, int sizeInBytesPerKVHead,
    int rewindDraftTokenCommonCount, int* rewindDraftTokenSeparateAdjustments, int const* seqSlotRemapping,
    hipStream_t stream)
{
    // make sure launch buffer is enough
    static_assert(MaxLayerCount * sizeof(KVCacheBuffer) <= 3072);
    if (seqCount == 0 || layerCount == 0)
    {
        return;
    }
    int alignedBytes = 16;
    while (alignedBytes > 0 && (sizeInBytesPerKVHead % alignedBytes != 0))
    {
        alignedBytes >>= 1;
    }
    TLLM_CHECK_WITH_INFO(alignedBytes > 0, "alignedByte should be positive");
    int eltCountPerHead = sizeInBytesPerKVHead / alignedBytes;
    dim3 grid(seqCount, numKVHeads, layerCount);
    dim3 block(128, 1, 1);
    std::array<KVCacheBuffer, MaxLayerCount> kvCacheBufferArray;
    for (int i = 0; i < layerCount; i++)
    {
        kvCacheBufferArray[i] = kvCacheBuffers[i];
    }
    void (*pKernelFunc)(std::array<KVCacheBuffer, MaxLayerCount>, int const*, IndexType const*, int32_t const*, int,
        int const*, int const*, int)
        = nullptr;
    switch (alignedBytes)
    {
    case 16:
    {
        pKernelFunc = &updateKVCacheDraftTokenLocationBatchedKernel<KVCacheBuffer, MaxLayerCount, int4>;
        break;
    }
    case 8:
    {
        pKernelFunc = &updateKVCacheDraftTokenLocationBatchedKernel<KVCacheBuffer, MaxLayerCount, int64_t>;
        break;
    }
    case 4:
    {
        pKernelFunc = &updateKVCacheDraftTokenLocationBatchedKernel<KVCacheBuffer, MaxLayerCount, int32_t>;
        break;
    }
    case 2:
    {
        pKernelFunc = &updateKVCacheDraftTokenLocationBatchedKernel<KVCacheBuffer, MaxLayerCount, int16_t>;
        break;
    }
    default:
    {
        TLLM_CHECK_WITH_INFO(alignedBytes == 1, "Strange alignedBytes");
        pKernelFunc = &updateKVCacheDraftTokenLocationBatchedKernel<KVCacheBuffer, MaxLayerCount, int8_t>;
        break;
    }
    }
    pKernelFunc<<<grid, block, 0, stream>>>(kvCacheBufferArray, seqAcceptedDraftTokenOffsets,
        packedAcceptedDraftTokensIndices, pastKeyValueLengths, rewindDraftTokenCommonCount,
        rewindDraftTokenSeparateAdjustments, seqSlotRemapping, eltCountPerHead);
    TLLM_CUDA_CHECK(hipGetLastError());
}

/*!
 * Update KV cache for parallel decoding algorithms.
 * In following examples, we assume we have 2 sequences, accepted count is [3, 2]
 * @tparam KVCacheBuffer : Type of KV cache, should be LinearKVCache or KVBlockArray
 * @param kvCacheBuffers : list of KVCacheBuffer object
 * @param seqAcceptedDraftTokenOffsets : Array of length seqCount + 1, like [0, 3, 5]
 * @param packedAcceptedDraftTokensIndices : Array of length seqAcceptedDraftTokenOffsets[seqCount], each value is in
 * range [0, maxDraftTokenCount - 1]
 * @param pastKeyValueLengths : Array of length seqCount, meaning how many tokens are already in KV cache
 * @param seqCount : Count of sequence
 * @param numKVHeads : Number of KV heads
 * @param sizeInBytesPerKVHead : Size of each KV head
 * @param rewindDraftTokenCommonCount : Common count to rewind
 * @param rewindDraftTokenSeparateAdjustments : Separate adjustment to rewind for each sequence, if nullptr, just use
 * rewindDraftTokenCommonCount, else use rewindDraftTokenSeparateAdjustments[i] + rewindDraftTokenCommonCount
 * @param stream : CUDA stream to use.
 */
template <typename KVCacheBuffer>
void updateKVCacheDraftTokenLocation(KVCacheBuffer const* kvCacheBuffers, int const* seqAcceptedDraftTokenOffsets,
    IndexType const* packedAcceptedDraftTokensIndices, int32_t const* pastKeyValueLengths, int layerCount, int seqCount,
    int numKVHeads, int sizeInBytesPerKVHead, int rewindDraftTokenCommonCount, int* rewindDraftTokenSeparateAdjustments,
    int const* seqSlotRemapping, hipStream_t stream)
{
    int startLayer = 0;
    static constexpr int kMaxLayersPerIter = 32;
    while (startLayer < layerCount)
    {
        int microBatchLayerCount = std::min(layerCount - startLayer, kMaxLayersPerIter);
        updateKVCacheDraftTokenLocationBatched<KVCacheBuffer, kMaxLayersPerIter>(kvCacheBuffers + startLayer,
            seqAcceptedDraftTokenOffsets, packedAcceptedDraftTokensIndices, pastKeyValueLengths, microBatchLayerCount,
            seqCount, numKVHeads, sizeInBytesPerKVHead, rewindDraftTokenCommonCount,
            rewindDraftTokenSeparateAdjustments, seqSlotRemapping, stream);
        startLayer += microBatchLayerCount;
    }
}

void updateLinearKVCacheDraftTokenLocation(int const* seqAcceptedDraftTokenOffsets,
    IndexType const* packedAcceptedDraftTokensIndices, int32_t const* pastKeyValueLengths,
    int8_t* const* pastKeyValueList, int layerCount, int seqCount, int numKVHeads, int sizeInBytesPerKVHead,
    int rewindDraftTokenCommonCount, int* rewindDraftTokenSeparateAdjustments, int const* seqSlotRemapping,
    int maxKVCacheLen, hipStream_t stream)
{
    std::vector<KVLinearBuffer> kvLinearBuffers;
    kvLinearBuffers.reserve(layerCount);
    auto const sizePerToken = numKVHeads * sizeInBytesPerKVHead;
    for (int i = 0; i < layerCount; i++)
    {
        kvLinearBuffers.emplace_back(
            seqCount, maxKVCacheLen, sizePerToken, maxKVCacheLen, 0, false, pastKeyValueList[i]);
    }
    updateKVCacheDraftTokenLocation(kvLinearBuffers.data(), seqAcceptedDraftTokenOffsets,
        packedAcceptedDraftTokensIndices, pastKeyValueLengths, layerCount, seqCount, numKVHeads, sizeInBytesPerKVHead,
        rewindDraftTokenCommonCount, rewindDraftTokenSeparateAdjustments, seqSlotRemapping, stream);
}

void updateKVBlockArrayDraftTokenLocation(int const* seqAcceptedDraftTokenOffsets,
    IndexType const* packedAcceptedDraftTokensIndices, int32_t const* pastKeyValueLengths, void* const* pointerArray,
    KVBlockArray::DataType* offsetArray, int layerCount, int seqCount, int numKVHeads, int sizeInBytesPerKVHead,
    int rewindDraftTokenCommonCount, int* rewindDraftTokenSeparateAdjustments, int const* seqSlotRemapping,
    int maxKVCacheLen, int maxBlocksPerSeq, int tokensPerBlock, hipStream_t stream)
{
    std::vector<KVBlockArray> kvBlockArrays;
    kvBlockArrays.reserve(layerCount);
    auto const bytesPerToken = numKVHeads * sizeInBytesPerKVHead;
    auto const bytesPerBlock = tokensPerBlock * bytesPerToken;
    for (int layerIdx = 0; layerIdx < layerCount; layerIdx++)
    {
        auto const layerOffset = layerIdx * 2 * bytesPerBlock;
        auto* const primaryPoolPointer
            = reinterpret_cast<void*>(reinterpret_cast<char*>(pointerArray[0]) + layerOffset);
        auto* const secondaryPoolPointer
            = reinterpret_cast<void*>(reinterpret_cast<char*>(pointerArray[1]) + layerOffset);

        kvBlockArrays.emplace_back(seqCount, maxBlocksPerSeq, tokensPerBlock, bytesPerToken, maxKVCacheLen, 0,
            primaryPoolPointer, secondaryPoolPointer, offsetArray);
    }
    updateKVCacheDraftTokenLocation(kvBlockArrays.data(), seqAcceptedDraftTokenOffsets,
        packedAcceptedDraftTokensIndices, pastKeyValueLengths, layerCount, seqCount, numKVHeads, sizeInBytesPerKVHead,
        rewindDraftTokenCommonCount, rewindDraftTokenSeparateAdjustments, seqSlotRemapping, stream);
}

void updateLinearKVCacheDraftTokenLocationCommonRewind(int const* seqAcceptedDraftTokenOffsets,
    IndexType const* packedAcceptedDraftTokensIndices, int32_t const* pastKeyValueLengths,
    int8_t* const* pastKeyValueList, int layerCount, int seqCount, int numKVHeads, int sizeInBytesPerKVHead,
    int rewindDraftTokenCount, int const* seqSlotRemapping, int maxKVCacheLen, hipStream_t stream)
{
    updateLinearKVCacheDraftTokenLocation(seqAcceptedDraftTokenOffsets, packedAcceptedDraftTokensIndices,
        pastKeyValueLengths, pastKeyValueList, layerCount, seqCount, numKVHeads, sizeInBytesPerKVHead,
        rewindDraftTokenCount, nullptr, seqSlotRemapping, maxKVCacheLen, stream);
}

void updateKVBlockArrayDraftTokenLocationCommonRewind(int const* seqAcceptedDraftTokenOffsets,
    IndexType const* packedAcceptedDraftTokensIndices, int32_t const* pastKeyValueLengths, void* const* pointerArray,
    KVBlockArray::DataType* offsetArray, int layerCount, int seqCount, int numKVHeads, int sizeInBytesPerKVHead,
    int rewindDraftTokenCount, int const* seqSlotRemapping, int maxKVCacheLen, int maxBlocksPerSeq, int tokensPerBlock,
    hipStream_t stream)
{
    updateKVBlockArrayDraftTokenLocation(seqAcceptedDraftTokenOffsets, packedAcceptedDraftTokensIndices,
        pastKeyValueLengths, pointerArray, offsetArray, layerCount, seqCount, numKVHeads, sizeInBytesPerKVHead,
        rewindDraftTokenCount, nullptr, seqSlotRemapping, maxKVCacheLen, maxBlocksPerSeq, tokensPerBlock, stream);
}

void updateLinearKVCacheDraftTokenLocationSeparateRewind(int const* seqAcceptedDraftTokenOffsets,
    IndexType const* packedAcceptedDraftTokensIndices, int32_t const* pastKeyValueLengths,
    int8_t* const* pastKeyValueList, int layerCount, int seqCount, int numKVHeads, int sizeInBytesPerKVHead,
    int* rewindDraftTokenCounts, int const* seqSlotRemapping, int maxKVCacheLen, hipStream_t stream)
{
    updateLinearKVCacheDraftTokenLocation(seqAcceptedDraftTokenOffsets, packedAcceptedDraftTokensIndices,
        pastKeyValueLengths, pastKeyValueList, layerCount, seqCount, numKVHeads, sizeInBytesPerKVHead, 0,
        rewindDraftTokenCounts, seqSlotRemapping, maxKVCacheLen, stream);
}

void updateKVBlockArrayDraftTokenLocationSeparateRewind(int const* seqAcceptedDraftTokenOffsets,
    IndexType const* packedAcceptedDraftTokensIndices, int32_t const* pastKeyValueLengths, void* const* pointerArray,
    KVBlockArray::DataType* offsetArray, int layerCount, int seqCount, int numKVHeads, int sizeInBytesPerKVHead,
    int* rewindDraftTokenCounts, int const* seqSlotRemapping, int maxKVCacheLen, int maxBlocksPerSeq,
    int tokensPerBlock, hipStream_t stream)
{
    updateKVBlockArrayDraftTokenLocation(seqAcceptedDraftTokenOffsets, packedAcceptedDraftTokensIndices,
        pastKeyValueLengths, pointerArray, offsetArray, layerCount, seqCount, numKVHeads, sizeInBytesPerKVHead, 0,
        rewindDraftTokenCounts, seqSlotRemapping, maxKVCacheLen, maxBlocksPerSeq, tokensPerBlock, stream);
}

} // namespace tensorrt_llm::kernels::parallel_decoding
