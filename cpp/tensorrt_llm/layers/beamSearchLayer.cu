#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/beamSearchKernels.h"
#include "tensorrt_llm/layers/beamSearchLayer.h"
#include "tensorrt_llm/layers/defaultDecodingParams.h"
#include "tensorrt_llm/layers/layerUtils.h"
#include <limits>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm
{
namespace layers
{

template <typename T>
BeamSearchLayer<T>::BeamSearchLayer(
    DecoderDomain const& decoderDomain, hipStream_t stream, std::shared_ptr<IAllocator> allocator)
    : BaseLayer(decoderDomain, stream, std::move(allocator))
    , mVocabSize(decoderDomain.getVocabSize())
    , mVocabSizePadded(decoderDomain.getVocabSizePadded())
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
}

template <typename T>
BeamSearchLayer<T>::~BeamSearchLayer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
}

template <typename T>
void BeamSearchLayer<T>::setup(runtime::SizeType32 const batch_size, runtime::SizeType32 const beam_width,
    runtime::SizeType32 const* batchSlots, std::shared_ptr<BaseSetupParams> baseSetupParams)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    TLLM_CHECK_WITH_INFO(
        beam_width <= nMaxBeamWidth, std::string("Beam width is larger than the maximum supported (64)."));

    auto setupParams = std::dynamic_pointer_cast<BeamSearchSetupParams>(baseSetupParams);

    mDiversityRateHost.resize(batch_size);
    mLengthPenaltyHost.resize(batch_size);
    mEarlyStoppingHost.resize(batch_size);
    allocateBuffer(batch_size, beam_width);

    auto constexpr fltMax = std::numeric_limits<float>::max();
    auto constexpr fltMin = std::numeric_limits<float>::lowest();
    auto constexpr fltEpsilon = std::numeric_limits<float>::epsilon();

    FillBuffers const fillBuffers{batch_size, batch_size, mStream};
    fillBuffers(setupParams->beam_search_diversity_rate, DefaultDecodingParams::getBeamSearchDiversity(),
        mDiversityRateHost, mDiversityRateDevice, (int*) nullptr, std::make_pair(-fltEpsilon, fltMax),
        "diveristy rate");
    fillBuffers(setupParams->length_penalty, DefaultDecodingParams::getLengthPenalty(), mLengthPenaltyHost,
        mLengthPenaltyDevice, (int*) nullptr, std::make_pair(fltMin, fltMax), "length penalty");
    fillBuffers(setupParams->early_stopping, DefaultDecodingParams::getEarlyStopping(), mEarlyStoppingHost,
        mEarlyStoppingDevice, (int*) nullptr, std::make_pair(fltMin, fltMax), "early stopping");
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

__global__ void updateIndirCacheKernel(int* tgt_indir_cache, int const* src_indir_cache, BeamHypotheses bh,
    int const* inputLengths, int max_attention_window, int sink_token_length)
{
    // Update indirections from steps `nInputLength[nBBId]` to step `sequence_lengths[nBBId]`
    int const time_step = threadIdx.x + blockIdx.x * blockDim.x;
    int const nBBId = blockIdx.y;
    int const nBM{bh.nBeamWidth};
    int const batch_id = nBBId / nBM;
    int const beam_id = nBBId % nBM;
    int const current_step{bh.sequenceLengths[nBBId] - 1}; // the sequence_lengths is updated, need to minus 1
    int const nInputLength{inputLengths == nullptr ? 0 : inputLengths[nBBId]};

    // Return early when the nBBId or timestep is out of the bound
    // No update for the indices of context part since KV Cache is shared and fixed for context part
    if (nBBId >= nBM * bh.nBatchSizeLocal || time_step >= bh.nMaxSeqLen || time_step < nInputLength
        || time_step < (bh.nMaxSeqLen - max_attention_window) || bh.finished[nBBId].isFinished())
    {
        return;
    }
    int time_step_circ = time_step;
    if (time_step_circ >= sink_token_length)
    {
        time_step_circ
            = sink_token_length + (time_step - sink_token_length) % (max_attention_window - sink_token_length);
    }

    // for the parent_ids, we will still keep it for all past tokens (i.e. bh.nMaxSeqLen)
    int const src_beam = bh.parentIdsPtr[batch_id][beam_id * bh.nMaxSeqLen + current_step];

    // for the indir tables, we have the cyclic kv cache.
    uint32_t const tgt_offset = batch_id * nBM * max_attention_window + beam_id * max_attention_window + time_step_circ;
    uint32_t const src_offset
        = batch_id * nBM * max_attention_window + src_beam * max_attention_window + time_step_circ;

    tgt_indir_cache[tgt_offset] = (time_step == current_step) ? beam_id : src_indir_cache[src_offset];
}

void updateIndirCacheKernelLauncher(int* tgt_cache_indirection, int const* src_cache_indirection, BeamHypotheses& bh,
    int const* inputLengths, int max_attention_window, int sink_token_length, hipStream_t stream)
{
    int const max_seq_len_aligned = (bh.nMaxSeqLen + 31) / 32;
    dim3 const grid(max_seq_len_aligned, bh.nBatchSizeLocal * bh.nBeamWidth);
    updateIndirCacheKernel<<<grid, 32, 0, stream>>>(
        tgt_cache_indirection, src_cache_indirection, bh, inputLengths, max_attention_window, sink_token_length);
}

template <typename T>
void BeamSearchLayer<T>::forward(
    std::shared_ptr<BaseOutputParams> baseOutputs, std::shared_ptr<BaseInputParams> baseInputs)
{
    TLLM_LOG_TRACE("%s", __PRETTY_FUNCTION__);

    auto fp = std::dynamic_pointer_cast<BeamSearchInputParams>(baseInputs);
    auto op = std::dynamic_pointer_cast<BeamSearchOutputParams>(baseOutputs);

    TLLM_CHECK_WITH_INFO(op->beamHypotheses, std::string("Output BeamHypotheses is not set."));
    TLLM_CHECK_WITH_INFO(op->sequence_length->template getPtr<int>() != nullptr || mLengthPenaltyDevice == nullptr,
        std::string("Current sequence lengths must be set for length penalty computation."));
    TLLM_CHECK_WITH_INFO(fp->ite == 0, "Pipeline Parallelism is not supported yet !");

    BeamHypotheses& bh{*op->beamHypotheses};
    bh.nBatchSize = static_cast<std::int32_t>(op->output_ids_ptr.shape[0]);
    bh.nBeamWidth = static_cast<std::int32_t>(op->output_ids_ptr.shape[1]);
    bh.nIte = fp->ite;
    bh.nBatchSizeLocal = fp->logits.shape[0];
    bh.nMaxSeqLen = static_cast<std::int32_t>(op->output_ids_ptr.shape[2]);
    bh.nVocabSize = mVocabSizePadded;
    bh.diversityRates = mDiversityRateDevice;
    bh.lengthPenalties = mLengthPenaltyDevice;
    bh.earlyStoppings = mEarlyStoppingDevice;
    // bh.inputLengths = (fp->input_lengths) ? fp->input_lengths->template getPtr<int const>() : nullptr;
    // TODO: unify the assignment of inputLengths
    bh.endIds = fp->end_ids.template getPtr<int const>();
    bh.logProbs = (op->output_log_probs) ? op->output_log_probs->template getPtr<float>() : nullptr;
    // TODO (wili): here is a error in C++ workflow
    // In Python workflow, `op.output_log_probs` here is assigned by `outputs.output_log_probs_tiled` [MSL, BS, BM]
    // (function layersForward in file cpp/tensorrt_llm/layers/dynamicDecodeLayer.cpp)
    // But in C++ workflow, `op.output_log_probs` here is assigned by `output.logProbs` [BS, BM, MSL]
    // (function prepareOutputs in file cpp/tensorrt_llm/runtime/gptDecoder.cpp)
    bh.sequenceLengths = op->sequence_length->template getPtr<int>();
    bh.cumLogProbs = op->cum_log_probs->template getPtr<float>();
    bh.finished = reinterpret_cast<FinishedState*>(op->finished->template getPtr<FinishedState::UnderlyingType>());
    bh.outputIdsPtr = op->output_ids_ptr.template getPtr<int*>();
    bh.parentIdsPtr = op->parent_ids_ptr.template getPtr<int*>();

    T const* logits = fp->logits.template getPtr<T>();
    T const* bias = static_cast<T const*>(nullptr);
    TLLM_CHECK_WITH_INFO(mWorkspaceSize >= 2 * bh.nBatchSize * bh.nBeamWidth * bh.nBeamWidth * 2,
        std::string("Workspace size is not enough for topk softmax."));

    invokeTopkSoftMax(logits, bias, mWorkspace, bh, mStream);
    sync_check_cuda_error();

    if (bh.nBeamWidth > 1)
    {
        auto* const inputLengths = fp->input_lengths ? fp->input_lengths->template getPtr<int const>() : nullptr;
        auto tgt_ci = op->tgt_cache_indirection.template getPtr<int>();
        auto src_ci = fp->src_cache_indirection.template getPtr<int const>();

        updateIndirCacheKernelLauncher(
            tgt_ci, src_ci, bh, inputLengths, fp->max_attention_window, fp->sink_token_length, mStream);
        sync_check_cuda_error();
    }
}

template <typename T>
void BeamSearchLayer<T>::allocateBuffer(runtime::SizeType32 const batch_size, runtime::SizeType32 const beam_width)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    int const nPadBeamWidth = padToNextPowerOfTwo(beam_width);
    // Unit of mWorkspaceSize is number of elements (not Byte), align to 4 for further optimization
    size_t nTopK = batch_size * nPadBeamWidth * nPadBeamWidth * 2;
    size_t nTempBuffer = batch_size * nPadBeamWidth * nMaxVocabPartForStage1FastKernel * (2 * (nPadBeamWidth * 2) + 2);
    mWorkspaceSize = roundUp(nTopK, 4) * 2 + roundUp(nTempBuffer, 4);
    mWorkspace = mAllocator->reMalloc(mWorkspace, sizeof(float) * mWorkspaceSize, true);
    mDiversityRateDevice = mAllocator->reMalloc(mDiversityRateDevice, sizeof(float) * batch_size, false);
    mLengthPenaltyDevice = mAllocator->reMalloc(mLengthPenaltyDevice, sizeof(float) * batch_size, false);
    mEarlyStoppingDevice = mAllocator->reMalloc(mEarlyStoppingDevice, sizeof(int) * batch_size, false);
    mIsAllocateBuffer = true;
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BeamSearchLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    if (mIsAllocateBuffer)
    {
        mAllocator->free((void**) (&mWorkspace));
        mAllocator->free((void**) (&mDiversityRateDevice));
        mAllocator->free((void**) (&mLengthPenaltyDevice));
        mAllocator->free((void**) (&mEarlyStoppingDevice));
        mIsAllocateBuffer = false;
    }
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template class BeamSearchLayer<float>;
template class BeamSearchLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
